#include "hip/hip_runtime.h"


// Includes
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <time.h>


// Constants used by the program
#define MAX_PITCH_VALUE_IN_BYTES       262144
#define MAX_TEXTURE_WIDTH_IN_BYTES     65536
#define MAX_TEXTURE_HEIGHT_IN_BYTES    32768
#define MAX_PART_OF_FREE_MEMORY_USED   0.9
#define BLOCK_DIM                      16


#ifndef max
#define max(a,b) (((a) (b)) ? (a) : (b))
#define min(a,b) (((a) < (b)) ? (a) : (b))
#endif


// Texture containing the reference points (if it is possible)
texture<float, 2, hipReadModeElementType> texA;

__global__ void cuComputeDistanceTexture(int wA, float * B, int wB, int pB, int dim, float* AB){
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if ( xIndex<wB && yIndex<wA ){
    float ssd = 0;
    for (int i=0; i<dim; i++){
      float tmp  = tex2D(texA, (float)yIndex, (float)i) - B[ i * pB + xIndex ];
      ssd += tmp * tmp;
    }
    AB[yIndex * pB + xIndex] = ssd;
  }
}

__global__ void cuComputeDistanceGlobal( float* A, int wA, int pA, float* B, int wB, int pB, int dim,  float* AB){

  // Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
  __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
  __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

  // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
  __shared__ int begin_A;
  __shared__ int begin_B;
  __shared__ int step_A;
  __shared__ int step_B;
  __shared__ int end_A;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Other variables
  float tmp;
  float ssd = 0;

  // Loop parameters
  begin_A = BLOCK_DIM * blockIdx.y;
  begin_B = BLOCK_DIM * blockIdx.x;
  step_A  = BLOCK_DIM * pA;
  step_B  = BLOCK_DIM * pB;
  end_A   = begin_A + (dim-1) * pA;

  // Conditions
  int cond0 = (begin_A + tx < wA); // used to write in shared memory
  int cond1 = (begin_B + tx < wB); // used to write in shared memory & to computations and to write in output matrix
  int cond2 = (begin_A + ty < wA); // used to computations and to write in output matrix

  // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
  for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {

    // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
    if (a/pA + ty < dim){
      shared_A[ty][tx] = (cond0)? A[a + pA * ty + tx] : 0;
      shared_B[ty][tx] = (cond1)? B[b + pB * ty + tx] : 0;
    }
    else{
      shared_A[ty][tx] = 0;
      shared_B[ty][tx] = 0;
    }

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
    if (cond2 && cond1){
      for (int k = 0; k < BLOCK_DIM; ++k){
        tmp = shared_A[k][ty] - shared_B[k][tx];
        ssd += tmp*tmp;
      }
    }

    // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory; each thread writes one element
  if (cond2 && cond1)
    AB[ (begin_A + ty) * pB + begin_B + tx ] = ssd;
}

__global__ void cuInsertionSort(float *dist, int dist_pitch, int *ind, int ind_pitch, int width, int height, int k){

  // Variables
  int l, i, j;
  float *p_dist;
  int   *p_ind;
  float curr_dist, max_dist;
  int   curr_row,  max_row;
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if (xIndex<width){

    // Pointer shift, initialization, and max value
    p_dist   = dist + xIndex;
    p_ind    = ind  + xIndex;
    max_dist = p_dist[0];
    p_ind[0] = 1;

    // Part 1 : sort kth firt elementZ
    for (l=1; l<k; l++){
      curr_row  = l * dist_pitch;
      curr_dist = p_dist[curr_row];
      if (curr_dist<max_dist){
        i=l-1;
        for (int a=0; a<l-1; a++){
          if (p_dist[a*dist_pitch]>curr_dist){
            i=a;
            break;
          }
        }
        for (j=l; j>i; j--){
          p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
          p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
        }
        p_dist[i*dist_pitch] = curr_dist;
        p_ind[i*ind_pitch]   = l+1;
      }
      else{
        p_ind[l*ind_pitch] = l+1;
      }
      max_dist = p_dist[curr_row];
    }

    // Part 2 : insert element in the k-th first lines
    max_row = (k-1)*dist_pitch;
    for (l=k; l<height; l++){
      curr_dist = p_dist[l*dist_pitch];
      if (curr_dist<max_dist){
        i=k-1;
        for (int a=0; a<k-1; a++){
          if (p_dist[a*dist_pitch]>curr_dist){
            i=a;
            break;
          }
        }
        for (j=k-1; j>i; j--){
          p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
          p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
        }
        p_dist[i*dist_pitch] = curr_dist;
        p_ind[i*ind_pitch]   = l+1;
        max_dist             = p_dist[max_row];
      }
    }
  }
}


__global__ void cuParallelSqrt(float *dist, int width, int pitch, int k){
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if (xIndex<width && yIndex<k)
    dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex]);
}



void printErrorMessage(hipError_t error, int memorySize){
  printf("==================================================\n");
  printf("MEMORY ALLOCATION ERROR  : %s\n", hipGetErrorString(error));
  printf("Whished allocated memory : %d\n", memorySize);
  printf("==================================================\n");
}



void knn_brute_force_garcia(float* ref_host, int ref_width, float* query_host, int query_width, int height, int k, float* dist_host, int* ind_host){

  unsigned int size_of_float = sizeof(float);
  unsigned int size_of_int   = sizeof(int);

  // Variables
  float        *query_dev;
  float        *ref_dev;
  float        *dist_dev;
  int          *ind_dev;
  hipArray    *ref_array;
  hipError_t  result;
  size_t       query_pitch;
  size_t       query_pitch_in_bytes;
  size_t       ref_pitch;
  size_t       ref_pitch_in_bytes;
  size_t       ind_pitch;
  size_t       ind_pitch_in_bytes;
  size_t       max_nb_query_traited;
  size_t       actual_nb_query_width;
  size_t       memory_total;
  size_t       memory_free;


  unsigned int use_texture = ( ref_width*size_of_float<=MAX_TEXTURE_WIDTH_IN_BYTES && height*size_of_float<=MAX_TEXTURE_HEIGHT_IN_BYTES );

  hipInit(0);

  hipCtx_t cuContext;
  hipDevice_t  cuDevice=0;
  hipCtxCreate(&cuContext, 0, cuDevice);
  hipMemGetInfo(&memory_free, &memory_total);
  cuCtxDetach (cuContext);
  max_nb_query_traited = ( memory_free * MAX_PART_OF_FREE_MEMORY_USED - size_of_float * ref_width*height ) / ( size_of_float * (height + ref_width) + size_of_int * k);
  max_nb_query_traited = min( query_width, (max_nb_query_traited / 16) * 16 );

  result = hipMallocPitch( (void **) &query_dev, &query_pitch_in_bytes, max_nb_query_traited * size_of_float, height + ref_width);
  if (result){
    printErrorMessage(result, max_nb_query_traited*size_of_float*(height+ref_width));
    return;
  }
  query_pitch = query_pitch_in_bytes/size_of_float;
  dist_dev    = query_dev + height * query_pitch;

  result = hipMallocPitch( (void **) &ind_dev, &ind_pitch_in_bytes, max_nb_query_traited * size_of_int, k);
  if (result){
    hipFree(query_dev);
    printErrorMessage(result, max_nb_query_traited*size_of_int*k);
    return;
  }
  ind_pitch = ind_pitch_in_bytes/size_of_int;

  if (use_texture){

    hipChannelFormatDesc channelDescA = hipCreateChannelDesc<float>();
    result = hipMallocArray( &ref_array, &channelDescA, ref_width, height );
    if (result){
      printf("ref_array\n");
      printErrorMessage(result, ref_width*height*size_of_float);
      hipFree(ind_dev);
      hipFree(query_dev);
      return;
    }
    hipMemcpyToArray( ref_array, 0, 0, ref_host, ref_width * height * size_of_float, hipMemcpyHostToDevice );

    texA.addressMode[0] = hipAddressModeClamp;
    texA.addressMode[1] = hipAddressModeClamp;
    texA.filterMode     = hipFilterModePoint;
    texA.normalized     = 0;
    hipBindTextureToArray(texA, ref_array);

  }
  else{

    // Allocation of global memory
    result = hipMallocPitch( (void **) &ref_dev, &ref_pitch_in_bytes, ref_width * size_of_float, height);
    if (result){
      printErrorMessage(result,  ref_width*size_of_float*height);
      hipFree(ind_dev);
      hipFree(query_dev);
      return;
    }
    ref_pitch = ref_pitch_in_bytes/size_of_float;
    hipMemcpy2D(ref_dev, ref_pitch_in_bytes, ref_host, ref_width*size_of_float,  ref_width*size_of_float, height, hipMemcpyHostToDevice);
  }

  // Split queries to fit in GPU memory
  for (int i=0; i<query_width; i+=max_nb_query_traited){

    // Number of query points considered
    actual_nb_query_width = min( max_nb_query_traited, query_width-i );

    // Copy of part of query actually being treated
    hipMemcpy2D(query_dev, query_pitch_in_bytes, &query_host[i], query_width*size_of_float, actual_nb_query_width*size_of_float, height, hipMemcpyHostToDevice);

    // Grids ans threads
    dim3 g_16x16(actual_nb_query_width/16, ref_width/16, 1);
    dim3 t_16x16(16, 16, 1);
    if (actual_nb_query_width%16 != 0){
     g_16x16.x += 1;
    }
    if (ref_width  %16 != 0){
     g_16x16.y += 1;
    }
    //
    dim3 g_256x1(actual_nb_query_width/256, 1, 1);
    dim3 t_256x1(256, 1, 1);
    if (actual_nb_query_width%256 != 0){
     g_256x1.x += 1;
    }
    //
    dim3 g_k_16x16(actual_nb_query_width/16, k/16, 1);
    dim3 t_k_16x16(16, 16, 1);
    if (actual_nb_query_width%16 != 0){
     g_k_16x16.x += 1;
    }
    if (k  %16 != 0){
     g_k_16x16.y += 1;
    }

    // Kernel 1: Compute all the distances
    if (use_texture){
      cuComputeDistanceTexture<<<g_16x16,t_16x16>>>(ref_width, query_dev, actual_nb_query_width, query_pitch, height, dist_dev);
    }
    else{
      cuComputeDistanceGlobal<<<g_16x16,t_16x16>>>(ref_dev, ref_width, ref_pitch, query_dev, actual_nb_query_width, query_pitch, height, dist_dev);
    }

    // Kernel 2: Sort each column
    cuInsertionSort<<<g_256x1,t_256x1>>>(dist_dev, query_pitch, ind_dev, ind_pitch, actual_nb_query_width, ref_width, k);

    // Kernel 3: Compute square root of k first elements
    cuParallelSqrt<<<g_k_16x16,t_k_16x16>>>(dist_dev, query_width, query_pitch, k);

    // Memory copy of output from device to host
    hipMemcpy2D(&dist_host[i], query_width*size_of_float, dist_dev, query_pitch_in_bytes, actual_nb_query_width*size_of_float, k, hipMemcpyDeviceToHost);
    hipMemcpy2D(&ind_host[i],  query_width*size_of_int,   ind_dev,  ind_pitch_in_bytes,   actual_nb_query_width*size_of_int,   k, hipMemcpyDeviceToHost);
  }

  // Free memory
  if (use_texture){
    hipFreeArray(ref_array);
  }
  else{
    hipFree(ref_dev);
    hipFree(ind_dev);
    hipFree(query_dev);
  }
}



int main(int argc, char const *argv[])
{

  float* ref;                 // Pointer to reference point array
  float* query;               // Pointer to query point array
  float* dist;                // Pointer to distance array
  int*   ind;                 // Pointer to index array
  int    ref_nb;              // Reference point number, max=65535
  int    query_nb;            // Query point number,     max=65535
  int    dim;                 // Dimension of points
  int    k          = 20;     // Nearest neighbors to consider
  int    iterations = 1;
  int    i;

  char fileName[] = "data/knn_brute_force_6553_ref_points_1_query_point.data";
  FILE* file;
  #ifdef WIN64
    fopen_s( &file, fileName, "rb" );
  #else
    file = fopen(fileName, "rb");
  #endif

  fread(&ref_nb, sizeof(int), 1, file);
  fread(&query_nb, sizeof(int), 1, file);
  fread(&dim, sizeof(int), 1, file);

  ref    = (float *) malloc(ref_nb   * dim * sizeof(float));
  query  = (float *) malloc(query_nb * dim * sizeof(float));
  dist   = (float *) malloc(query_nb * k * sizeof(float));
  ind    = (int *)   malloc(query_nb * k * sizeof(int));

  for (int count = 0; count < ref_nb*dim; count++)
  {
    fread(&ref[count], sizeof(float), 1, file);
  }
  for (int count = 0; count < query_nb*dim; count++)
  {
    fread(&query[count], sizeof(float), 1, file);
  }

  fclose(file);
     hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   float elapsed_time;
hipEventRecord(start, 0);
    // Display informations
   printf("Number of reference points      : %6d\n", ref_nb  );
   printf("Dimension of points             : %4d\n", dim     );
   printf("Number of neighbors to consider : %4d\n", k       );
   printf("Processing kNN search           :"                );

  for (i=0; i<iterations; i++){
    knn_brute_force_garcia(ref, ref_nb, query, query_nb, dim, k, dist, ind);
  }
  
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);
  printf(" done in %f s \n", elapsed_time/1000);
    printf("\nFound indexes: [");
  for (int i = 0; i < k; ++i)
  {
    printf("%d, ",ind[i] );
  }
  printf("]\n");


  printf("Distances: [");
  for (int i = 0; i < k; ++i)
  {
    printf("%f, ",dist[i]);
  }
  printf("]\n");
  // Destroy cuda event object and free memory
  hipEventDestroy(start);
  hipEventDestroy(stop);
  int correct_ind[] = {3261, 2799, 5752, 1837, 522, 5065, 5410, 1915, 2618, 627, 6095, 305, 3375, 269, 6180, 4963, 2216, 3393, 31, 5061};
  int pass = 1;
  for (int i = 0; i < k; ++i)
  {
    if(ind[i] != correct_ind[i]){
      pass=0;
    }
  }

  if (pass == 1)
  {
    printf("PASS: True\n");
  }
  else{
    printf("PASS: False\n");

  }

  free(ind);
  free(dist);
  free(query);
  free(ref);

  return 0;
}