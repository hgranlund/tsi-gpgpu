#include "test-common.cuh"

void populatePoints(struct Point *points, int n)
{
    int i;
    float temp;
    srand(time(NULL));

    for (i = 0; i < n; ++i)
    {
        struct Point t;
        temp = n - i - 1;

        t.p[0] = temp, t.p[1] = temp, t.p[2] = temp;

        points[i] = t;
    }
}

void populatePointSs(struct PointS *points, int n)
{
    int i;
    float temp;
    srand(time(NULL));

    for (i = 0; i < n; ++i)
    {
        struct PointS t;
        temp = n - i - 1;

        t.p[0] = temp, t.p[1] = temp, t.p[2] = temp;

        points[i] = t;
    }
}

void cudaStartTiming(hipEvent_t &start, hipEvent_t &stop, float &elapsed_time)
{
    elapsed_time = 0;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start, 0));
}

void cudaStopTiming(hipEvent_t &start, hipEvent_t &stop, float &elapsed_time)
{
    checkCudaErrors(hipEventRecord(stop, 0));
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
}

void printCudaTiming(float elapsed_time, float bytes, int n)
{
    double throughput = 1.0e-9 * ((double)bytes) / (elapsed_time * 1e-3);

    printf("Throughput = %.4f GB/s, Time = %.5f ms, Size = %u Elements\n", throughput, elapsed_time, n);
}

void ASSERT_TREE_EQ(struct Point *expected_tree, struct Point *actual_tree, int n)
{
    int i, j;

    for (i = 0; i < n; ++i)
    {
        for (j = 0; j < 3; ++j)
        {
            ASSERT_EQ(expected_tree[i].p[j] , actual_tree[i].p[j]) << "failed with i = " << i << " j = " << j ;
        }
    }
}

void ASSERT_TREE_LEVEL_OK(PointS *points, int *steps, int n, int p)
{
    struct PointS *t_points;

    for (int i = 0; i < p; ++i)
    {
        t_points = points + steps[i * 2];
        n =  steps[i * 2 + 1] - steps[i * 2];

        for (int i = 0; i < n / 2; ++i)
        {
            ASSERT_LE(t_points[i].p[0], t_points[n / 2].p[0]) << "Faild with n = " << n << " and p " << p;
        }

        for (int i = n / 2; i < n; ++i)
        {
            ASSERT_GE(t_points[i].p[0], t_points[n / 2].p[0]) << "Faild with n = " << n << " and p " << p;
        }
    }
}
