

// Includes
#include <kNN-brute-force-bitonic.cuh>
#include <kNN-brute-force-reduce.cuh>
#include <knn_gpgpu.h>
#include <stdio.h>

#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <assert.h>
#include "hip/hip_runtime_api.h"


#define SHARED_SIZE_LIMIT 1024U
#define checkCudaErrors(val)           check ( (val), #val, __FILE__, __LINE__ )

void  run_iteration(int ref_nb, int k, int iterations)
{
    float *ref;
    float *query;
    float *dist;
    int   *ind;
    int    query_nb     = 1;
    int    dim        = 3;
    int    i;
    ref    = (float *) malloc(ref_nb   * dim * sizeof(float));
    query  = (float *) malloc(query_nb * dim * sizeof(float));
    dist   = (float *) malloc( k * sizeof(float));
    ind    = (int *)   malloc( k * sizeof(int));

    srand ( (unsigned int)time(NULL) );
    for (i = 0 ; i < ref_nb   * dim ; i++)
    {
        ref[i]    = (float)rand() / (float)1000;
    }
    for (i = 0 ; i < query_nb * dim ; i++)
    {
        query[i]  = (float)rand() / (float)1000;
    }


    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float elapsed_time = 0;

    checkCudaErrors(hipEventRecord(start, 0));

    for (int i = 0; i < iterations; ++i)
    {
        knn_brute_force_bitonic(ref, ref_nb, query, dim, k, dist, ind);

        // knn_brute_force_reduce(ref, ref_nb, query, dim, k, dist, ind);
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("%d, %d, %f \n", k, ref_nb, elapsed_time / iterations);

    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    free(ind);
    free(dist);
    free(query);
    free(ref);
}

int main(int argc, char const *argv[])
{

    printf("Running Knn-brute-force with no memory optimalisations\n");
    printf("k, n, time(ms) \n");
    for (int i = 10000000; i <= 10000000; i <<= 1)
    {
        hipDeviceSynchronize();
        hipDeviceReset();
        run_iteration(i, 1, 5);
    }
}
