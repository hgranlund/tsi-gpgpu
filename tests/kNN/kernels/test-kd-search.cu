#include "hip/hip_runtime.h"
#include <kd-tree-naive.cuh>
#include <kd-search.cuh>
#include <knn_gpgpu.h>
#include <point.h>

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "gtest/gtest.h"


#define debug 0

#define checkCudaErrors(val)           check ( (val), #val, __FILE__, __LINE__ )
#define FILE (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define debugf(fmt, ...) if(debug)printf("%s:%d: " fmt, FILE, __LINE__, __VA_ARGS__);



// void ASSERT_QUERY_EQ(struct Point *tree, int n, float qx, float qy, float qz, float ex, float ey, float ez)
// {

//     float query_point[3];
//     query_point[0] = qx, query_point[1] = qy, query_point[2] = qz;

//     int mid = (int) floor((n) / 2);

//     queryAll(tree, tree, n_tree, n_tree, 1, *h_result)

//     int best_fit = nn(query_point, tree, 0, mid);

//     float actual = tree[best_fit].p[0] + tree[best_fit].p[1] + tree[best_fit].p[2];
//     float expected = ex + ey + ez;

//     ASSERT_EQ(actual, expected);
// }

TEST(kd_search, kd_search_wiki_correctness)
{
    int wn = 6, k = 1;
    struct PointS *wiki = (PointS *) malloc(wn  * sizeof(PointS));
    struct Point *wiki_out = (Point *) malloc(wn  * sizeof(Point));
    int *result =  (int *) malloc(wn * k * sizeof(int));

    // (2,3), (5,4), (9,6), (4,7), (8,1), (7,2).
    wiki[0].p[0] = 2, wiki[0].p[1] = 3, wiki[0].p[2] = 0;
    wiki[1].p[0] = 5, wiki[1].p[1] = 4, wiki[1].p[2] = 0;
    wiki[2].p[0] = 9, wiki[2].p[1] = 6, wiki[2].p[2] = 0;
    wiki[3].p[0] = 4, wiki[3].p[1] = 7, wiki[3].p[2] = 0;
    wiki[4].p[0] = 8, wiki[4].p[1] = 1, wiki[4].p[2] = 0;
    wiki[5].p[0] = 7, wiki[5].p[1] = 2, wiki[5].p[2] = 0;

    hipDeviceReset();
    build_kd_tree(wiki, wn, wiki_out);
    store_locations(wiki_out, 0, wn, wn);
    queryAll(wiki_out, wiki_out, wn, wn, 1, result);

    // for (int i = 0; i < wn; ++i)
    // {
    // printf("%d, ", result[i]);
    // printf("(%3.1f, %3.1f, %3.1f)\n", wiki_out[result[i]].p[0], wiki_out[result[i]].p[1], wiki_out[result[i]].p[2]);
    // }
    // printf("\n");
    free(wiki);
    free(wiki_out);
    // ASSERT_QUERY_EQ(wiki_out, wn, 2, 3, 0, 2, 3, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 5, 4, 0, 5, 4, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 9, 6, 0, 9, 6, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 4, 7, 0, 4, 7, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 8, 1, 0, 8, 1, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 7, 2, 0, 7, 2, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 10, 10, 0, 9, 6, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 0, 0, 0, 2, 3, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 4, 4, 0, 5, 4, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 3, 2, 0, 2, 3, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 2, 6, 0, 4, 7, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 10, 0, 0, 8, 1, 0);
    // ASSERT_QUERY_EQ(wiki_out, wn, 0, 10, 0, 4, 7, 0);
}

TEST(kd_search, kd_search_timing)
{
    int i, n, k = 1;


    for (n = 1000000; n <= 1000000; n += 1000000)
    {
        PointS *points = (PointS *) malloc(n  * sizeof(PointS));
        Point *points_out = (Point *) malloc(n  * sizeof(Point));
        srand(time(NULL));

        for (i = 0; i < n; ++i)
        {
            PointS t;
            t.p[0] = rand() % 1000;
            t.p[1] = rand() % 1000;
            t.p[2] = rand() % 1000;
            points[i] = t;
        }


        build_kd_tree(points, n, points_out);



        store_locations(points_out, 0, n, n);

        int test_runs = 1;
        Point *query_data = (Point *) malloc(test_runs * sizeof(Point));
        int *result = (int *) malloc(test_runs * k * sizeof(int));

        for (i = 0; i < test_runs; i++)
        {
            Point point;
            point.p[0] = rand() % 1000;
            point.p[1] = rand() % 1000;
            point.p[2] = rand() % 1000;
            query_data[i] = point;
        }
        // printf("Build finish...\n");
        hipDeviceReset();
        hipEvent_t start, stop;
        unsigned int bytes = n * (sizeof(Point));
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));
        float elapsed_time = 0;

        checkCudaErrors(hipEventRecord(start, 0));

        queryAll(query_data, points_out, test_runs, n, k, result);

        checkCudaErrors(hipEventRecord(stop, 0));
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        elapsed_time = elapsed_time;
        double throughput = 1.0e-9 * ((double)bytes) / (elapsed_time * 1e-3);
        debugf("Search n query points, throughput = %.4f GB/s, time = %.5f ms, n = %u elements\n", throughput, elapsed_time, n);

        free(query_data);
        free(points_out);
        free(points);
        hipDeviceReset();
    };
};

// TEST(kd_search, kd_search_all_points)
// {
//     int i,
//         n = 100000,
//         n_qp = n,
//         k = 1,
//         *result;
//     Point *points;
//     points = (Point *) malloc(n  * sizeof(Point));
//     result = (int *) malloc(n_qp  * k * sizeof(int));
//     srand(time(NULL));

//     for (i = 0; i < n; ++i)
//     {
//         Point t;
//         t.p[0] = rand();
//         t.p[1] = rand();
//         t.p[2] = rand();
//         points[i] = t;
//     }

//     hipDeviceReset();
//     build_kd_tree(points, n);
//     store_locations(points, 0, n, n);

//     hipDeviceReset();
//     hipEvent_t start, stop;
//     unsigned int bytes = n * (sizeof(Point));
//     checkCudaErrors(hipEventCreate(&start));
//     checkCudaErrors(hipEventCreate(&stop));
//     float elapsed_time = 0;

//     checkCudaErrors(hipEventRecord(start, 0));

//     queryAll(points, points, n, n, k, result);

//     checkCudaErrors(hipEventRecord(stop, 0));
//     hipEventSynchronize(start);
//     hipEventSynchronize(stop);
//     hipEventElapsedTime(&elapsed_time, start, stop);
//     elapsed_time = elapsed_time;
//     double throughput = 1.0e-9 * ((double)bytes) / (elapsed_time * 1e-3);

//     printf("Searched for n queries, throughput = %.4f GB/s, time = %.5f ms, n = %u elements\n", throughput, elapsed_time, n);

//     free(points);
// }
