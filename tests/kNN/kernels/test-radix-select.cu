// Includes
#include <math.h>

#include <radix-select.cuh>
#include <knn_gpgpu.h>
#include "test-common.cuh"

#define debug 0
#define FILE (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define debugf(fmt, ...) if(debug)printf("%s:%d: " fmt, FILE, __LINE__, __VA_ARGS__);


int cpu_partition1(struct PointS *data, int l, int u, int bit)
{
    unsigned int radix = (1 << (31 - bit));
    struct PointS *temp = (struct PointS *)malloc(((u - l) + 1) * sizeof(PointS));
    int pos = 0;
    for (int i = l; i <= u; i++)
    {
        if (((*(int *) & (data[i].p[0]))&radix))
        {
            temp[pos++] = data[i];
        }
    }
    int result = u - pos;
    for (int i = l; i <= u; i++)
    {
        if (!((*(int *) & (data[i]))&radix))
        {
            temp[pos++] = data[i];
        }
    }
    pos = 0;
    for (int i = u; i >= l; i--)
    {
        data[i] = temp[pos++];
    }

    free(temp);
    return result;
}

struct PointS cpu_radixselect1(struct PointS *data, int l, int u, int m, int bit)
{
    struct PointS t;
    t.p[0] = 0;
    t.p[1] = 0;
    t.p[2] = 0;
    if (l == u) return (data[l]);
    if (bit > 32)
    {
        // debugf("cpu_radixselect1 fail!\n");
        return t;
    }
    int s = cpu_partition1(data, l, u, bit);
    if (s >= m) return cpu_radixselect1(data, l, s, m, bit + 1);
    return cpu_radixselect1(data, s + 1, u, m, bit + 1);
}

void printPoints1(struct PointS *l, int n)
{
    int i;
    if (debug)
    {
        // printf("[(%3.1f, %3.1f, %3.1f)", l[0].p[0], l[0].p[1], l[0].p[2]);
        printf("[%3.1f, ", l[0].p[0]);
        for (i = 1; i < n; ++i)
        {
            printf(", %3.1f, ", l[i].p[0]);
            // printf(", (%3.1f, %3.1f, %3.1f)", l[i].p[0], l[i].p[1], l[i].p[2]);
        }
        printf("]\n");
    }
}

__device__ __host__
unsigned int nextPowerOf22(unsigned int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

__device__ __host__
bool isPowTwo2(unsigned int x)
{
    return ((x & (x - 1)) == 0);
}

__device__ __host__
unsigned int prevPowerOf22(unsigned int n)
{
    if (isPowTwo2(n))
    {
        return n;
    }
    n = nextPowerOf22(n);
    return n >>= 1;

}

TEST(radix_selection, correctness)
{
    struct PointS *h_points;
    int n, dim = 0;
    for (n = 1000000; n <= 1000000; n += 1000000)
    {
        h_points = (struct PointS *) malloc(n * sizeof(PointS));

        if (n > 10000)
        {
            populatePointSRosetta(h_points, n);
            // readPoints("/home/simenhg/workspace/tsi-gpgpu/tests/data/100_mill_points.data", n, h_points);
        }
        else
        {

            readPoints("/home/simenhg/workspace/tsi-gpgpu/tests/data/10000_points.data", n, h_points);
            populatePointSRosetta(h_points, n);
        }

        // printPoints1(h_points, n);

        struct PointS *d_points, *d_temp;
        int *partition;

        checkCudaErrors(
            hipMalloc((void **)&d_points, n * sizeof(PointS)));
        checkCudaErrors(
            hipMalloc((void **)&d_temp, n * sizeof(PointS)));
        checkCudaErrors(
            hipMalloc((void **)&partition, n * sizeof(int)));
        checkCudaErrors(
            hipMemcpy(d_points, h_points, n * sizeof(PointS), hipMemcpyHostToDevice));

        struct PointS cpu_result = cpu_radixselect1(h_points, 0, n - 1, n / 2, 0);

        radixSelectAndPartition(d_points, d_temp, partition, n, dim);

        checkCudaErrors(
            hipMemcpy(h_points, d_points, n * sizeof(PointS), hipMemcpyDeviceToHost));


        debugf("result_gpu = (%3.1f, %3.1f, %3.1f)\n", h_points[n / 2].p[0], h_points[n / 2].p[1], h_points[n / 2].p[2] );
        debugf("result_cpu = (%3.1f, %3.1f, %3.1f)\n", cpu_result.p[0], cpu_result.p[1], cpu_result.p[2] );
        // ASSERT_EQ(cpu_result.p[0], h_points[n / 2].p[0]) << "Faild with n = " << n;
        // ASSERT_EQ(cpu_result.p[1], h_points[n / 2].p[1]) << "Faild with n = " << n;
        // ASSERT_EQ(cpu_result.p[2], h_points[n / 2].p[2]) << "Faild with n = " << n;


        int *h_steps = (int *) malloc( 2 * sizeof(int));
        h_steps[0] = 0;
        h_steps[1] = n;

        ASSERT_TREE_LEVEL_OK(h_points, h_steps, n, 1, dim);
        // printPoints1(h_points, n);

        checkCudaErrors(
            hipFree(d_points));
        checkCudaErrors(
            hipFree(d_temp));
        checkCudaErrors(
            hipFree(partition));
        hipDeviceSynchronize();
        hipDeviceReset();
    }
}

TEST(radix_selection, timing)
{
    struct PointS *h_points;
    int n;

    for (n = 8388608; n <= 8388608; n <<= 1)
    {
        h_points = (struct PointS *) malloc(n * sizeof(PointS));

        populatePointSRosetta(h_points, n);
        // readPoints("/home/simenhg/workspace/tsi-gpgpu/tests/data/10000_points.data", n, h_points);

        struct PointS *d_points, *d_temp;
        int *partition;

        checkCudaErrors(
            hipMalloc((void **)&d_points, n * sizeof(PointS)));
        checkCudaErrors(
            hipMalloc((void **)&d_temp, n * sizeof(PointS)));
        checkCudaErrors(
            hipMalloc((void **)&partition, n * sizeof(int)));
        checkCudaErrors(
            hipMemcpy(d_points, h_points, n * sizeof(PointS), hipMemcpyHostToDevice));

        radixSelectAndPartition(d_points, d_temp, partition, n, 0);

        checkCudaErrors(
            hipMemcpy(h_points, d_points, n * sizeof(PointS), hipMemcpyDeviceToHost));

        float elapsed_time = 0;
        hipEvent_t start, stop;
        cudaStartTiming(start, stop, elapsed_time);

        checkCudaErrors(hipMemcpy(d_points, h_points, n  * sizeof(PointS), hipMemcpyHostToDevice));
        radixSelectAndPartition(d_points, d_temp, partition, n, 0);

        cudaStopTiming(start, stop, elapsed_time);

        int bytes = n * (sizeof(float)) ;
        printCudaTiming(elapsed_time, bytes, n);

        checkCudaErrors(
            hipMemcpy(h_points, d_points, n * sizeof(PointS), hipMemcpyDeviceToHost));

        checkCudaErrors(
            hipFree(d_points));
        checkCudaErrors(
            hipFree(partition));
        checkCudaErrors(
            hipFree(d_temp));
        hipDeviceSynchronize();
        hipDeviceReset();
    }
}


