// Includes
#include <kd-tree-naive.cuh>
#include <knn_gpgpu.h>
#include <stdio.h>
#include <gtest/gtest.h>

#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <assert.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define checkCudaErrors(val)           check ( (val), #val, __FILE__, __LINE__ )
#define inf 0x7f800000

#define debug 0
#define FILE (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define debugf(fmt, ...) if(debug)printf("%s:%d: " fmt, FILE, __LINE__, __VA_ARGS__);


    float cpu_partition(Point *data, int l, int u, int bit)
    {
      unsigned int radix=(1 << 31-bit);
      Point *temp = (Point *)malloc(((u-l)+1)*sizeof(Point));
      int pos = 0;
      for (int i = l; i<=u; i++)
      {
        if(((*(int*)&(data[i].p[0]))&radix))
        {
          temp[pos++] = data[i];
        }
      }
      int result = u-pos;
      for (int i = l; i<=u; i++)
      {
        if(!((*(int*)&(data[i]))&radix))
        {
          temp[pos++] = data[i];
        }
      }
      pos = 0;
      for (int i = u; i>=l; i--)
      {
        data[i] = temp[pos++];
      }

      free(temp);
      return result;
    }

    Point cpu_radixselect(Point *data, int l, int u, int m, int bit){

      if (l == u) return(data[l]);
      if (bit > 32) {printf("cpu_radixselect fail!\n"); return (Point){0,0,0};}
      int s = cpu_partition(data, l, u, bit);
      if (s>=m) return cpu_radixselect(data, l, s, m, bit+1);
      return cpu_radixselect(data, s+1, u, m, bit+1);
    }






void printPoints(Point* l, int n){
  int i;
  if (debug)
  {
    // printf("[(%3.1f, %3.1f, %3.1f)", l[0].p[0], l[0].p[1], l[0].p[2]);
    printf("[%3.1f, ", l[0].p[0]);
      for (i = 1; i < n; ++i)
      {
        printf(", %3.1f, ", l[i].p[0]);
        // printf(", (%3.1f, %3.1f, %3.1f)", l[i].p[0], l[i].p[1], l[i].p[2]);
      }
      printf("]\n");
    }
  }


  TEST(kernels, radix_selection){
    Point *h_points;
    float temp;
    unsigned int i,n;
    for (n = 4; n <=2000; n<<=1)
    {
      h_points = (Point*) malloc(n*sizeof(Point));
      srand ( (unsigned int)time(NULL) );
      for (i=0 ; i<n; i++)
      {
        temp =  (float) rand()/100000000;
        h_points[i]    = (Point) {temp, temp, temp};
      }

      printPoints(h_points,n);

      Point *d_points, *d_temp, *d_result, h_result;
      int *partition;
      checkCudaErrors(
        hipMalloc((void **)&d_result, sizeof(Point)));
      checkCudaErrors(
        hipMalloc((void **)&d_points, n*sizeof(Point)));
      checkCudaErrors(
        hipMalloc((void **)&d_temp, n*sizeof(Point)));
      checkCudaErrors(
        hipMalloc((void **)&partition, n*sizeof(int)));
      checkCudaErrors(
        hipMemcpy(d_points, h_points, n*sizeof(Point), hipMemcpyHostToDevice));


      Point cpu_result = cpu_radixselect(h_points, 0, n-1, n/2, 0);

      cuRadixSelectGlobal<<<1,2>>>(d_points, d_temp, n/2, n, partition, 0, d_result);
      checkCudaErrors(
       hipMemcpy(&h_result, d_result, sizeof(Point), hipMemcpyDeviceToHost));

      checkCudaErrors(
        hipMemcpy(h_points, d_points, n*sizeof(Point), hipMemcpyDeviceToHost));

      printPoints(h_points,n);

      debugf("result = (%3.1f, %3.1f, %3.1f)\n", h_points[n/2].p[0], h_points[n/2].p[1], h_points[n/2].p[2] );
      ASSERT_EQ(cpu_result.p[0], h_points[n/2].p[0]) << "Faild with n = " << n;
      ASSERT_EQ(cpu_result.p[1], h_points[n/2].p[1]) << "Faild with n = " << n;
      ASSERT_EQ(cpu_result.p[2], h_points[n/2].p[2]) << "Faild with n = " << n;

      for (int i = 0; i < n/2; ++i)
      {
        ASSERT_LE(h_points[i].p[0], h_points[n/2].p[0]) << "Faild with n = " << n;
        /* code */
      }
      for (int i = n/2; i < n; ++i)
      {
        ASSERT_GE(h_points[i].p[0], h_points[n/2].p[0]) << "Faild with n = " << n;
        /* code */
      }
        // printDistArray(h_points,n);
      checkCudaErrors(
        hipFree(d_points));
      checkCudaErrors(
        hipFree(partition));
      checkCudaErrors(
        hipFree(d_result));
      free(h_points);
      hipDeviceSynchronize();
      hipDeviceReset();
    }
  }

  TEST(kernels, radix_selection_time){
    Point *h_points;
    unsigned int i,n;
    n = 160000;

    h_points = (Point*) malloc(n*sizeof(Point));

    float temp;

    h_points = (Point*) malloc(n*sizeof(Point));
    srand ( (unsigned int)time(NULL) );
    for (i=0 ; i<n; i++)
    {
      temp =  (float) rand()/100000000;
      h_points[i]    = (Point) {temp, temp, temp};
    }

    printPoints(h_points,n);

    Point *d_points, *d_temp, *d_result, h_result;
    int *partition;
    checkCudaErrors(
      hipMalloc((void **)&d_result, sizeof(Point)));
    checkCudaErrors(
      hipMalloc((void **)&d_points, n*sizeof(Point)));
    checkCudaErrors(
      hipMalloc((void **)&d_temp, n*sizeof(Point)));
    checkCudaErrors(
      hipMalloc((void **)&partition, n*sizeof(int)));
    checkCudaErrors(
      hipMemcpy(d_points, h_points, n*sizeof(Point), hipMemcpyHostToDevice));



    hipEvent_t start, stop;
    unsigned int bytes = n * (sizeof(float)) ;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float elapsed_time=0;

    checkCudaErrors(hipEventRecord(start, 0));

    cuRadixSelectGlobal<<<1,1024>>>(d_points, d_temp, n/2, n, partition, 0, d_result);


    checkCudaErrors(hipEventRecord(stop, 0));
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    elapsed_time = elapsed_time ;
    double throughput = 1.0e-9 * ((double)bytes)/(elapsed_time* 1e-3);
    printf("radix-select, Throughput = %.4f GB/s, Time = %.5f ms, Size = %u Elements, NumDevsUsed = %d\n",
     throughput, elapsed_time, n, 1);

    checkCudaErrors(
     hipMemcpy(&h_result, d_result, sizeof(Point), hipMemcpyDeviceToHost));

    checkCudaErrors(
      hipMemcpy(h_points, d_points, n*sizeof(Point), hipMemcpyDeviceToHost));

    printPoints(h_points,n);

    checkCudaErrors(
      hipFree(d_points));
    checkCudaErrors(
      hipFree(partition));
    checkCudaErrors(
      hipFree(d_result));
    free(h_points);
    hipDeviceSynchronize();
    hipDeviceReset();
  }

