// Includes
#include <quick-select.cuh>
#include <kd-tree-naive.cuh>
#include <knn_gpgpu.h>
#include <stdio.h>
#include <gtest/gtest.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <assert.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define inf 0x7f800000
#define THREADS_PER_BLOCK 1024U
#define MAX_BLOCK_DIM_SIZE 65535U

#define debug 0

#define checkCudaErrors(val)           check ( (val), #val, __FILE__, __LINE__ )
#define FILE (strrchr(__FILE__, '/') ? strrchr(__FILE__, '/') + 1 : __FILE__)
#define debugf(fmt, ...) if(debug)printf("%s:%d: " fmt, FILE, __LINE__, __VA_ARGS__);



__host__  void h_printPointsArray_(Point *l, int n, char *s, int l_debug = 0)
{
    if (debug || l_debug)
    {
        printf("%10s: [ ", s);
        for (int i = 0; i < n; ++i)
        {
            printf("%3.1f, ", l[i].p[0]);
        }
        printf("]\n");
    }
}

TEST(kernels, quick_selection_shared)
{
    Point *h_points, *d_points;
    int numBlocks, numThreads;
    float temp;
    unsigned int i, n, p;
    for (n = 4; n <= 1000; n <<= 1)
    {
        p = 4;
        h_points = (Point *) malloc(n * p * sizeof(Point));
        srand ( (unsigned int)time(NULL) );
        for (i = 0 ; i < n * p; i++)
        {
            temp =  (float) rand() / 100000000;
            Point t;
            t.p[0] = temp;
            t.p[1] = temp;
            t.p[2] = temp;
            h_points[i]    = t;
        }
        getThreadAndBlockCountForQuickSelect(n, p, numBlocks, numThreads);

        checkCudaErrors(
            hipMalloc((void **)&d_points, n * p * sizeof(Point)));
        checkCudaErrors(
            hipMemcpy(d_points, h_points, n * p * sizeof(Point), hipMemcpyHostToDevice));

        h_printPointsArray_(h_points, n * p, "h_points", 0);

        quickSelectAndPartition(d_points, n , p, 0);

        checkCudaErrors(
            hipMemcpy(h_points, d_points, n * p * sizeof(Point), hipMemcpyDeviceToHost));

        h_printPointsArray_(h_points, n * p, "h_points after", 0);

        Point *t_points;
        int nn = n;
        for (int i = 0; i < p; ++i)
        {
            t_points = h_points + i * (1 + n);
            nn =  n - i;
            for (int i = 0; i < nn / 2; ++i)
            {
                ASSERT_LE(t_points[i].p[0], t_points[nn / 2].p[0]) << "Faild with n = " << nn << " and p " << p;
            }
            for (int i = n / 2; i < nn; ++i)
            {
                ASSERT_GE(t_points[i].p[0], t_points[nn / 2].p[0]) << "Faild with n = " << nn << " and p " << p;
            }
        }

        checkCudaErrors(
            hipFree(d_points));
        free(h_points);
        hipDeviceSynchronize();
        hipDeviceReset();
    }
}

TEST(kernels, quick_selection_time)
{
    Point *h_points, *d_points;
    int numBlocks, numThreads;
    float temp;
    unsigned int i, n, p;
    for (n = 2048; n <= 2048; n <<= 1)
    {
        p = 64;
        h_points = (Point *) malloc(n * p * sizeof(Point));
        srand ( (unsigned int)time(NULL) );
        for (i = 0 ; i < n * p; i++)
        {
            temp =  (float) rand() / 100000000;
            Point t;
            t.p[0] = temp;
            t.p[1] = temp;
            t.p[2] = temp;
            h_points[i]    = t;
        }
        getThreadAndBlockCountForQuickSelect(n, p, numBlocks, numThreads);

        checkCudaErrors(
            hipMalloc((void **)&d_points, n * p * sizeof(Point)));
        checkCudaErrors(
            hipMemcpy(d_points, h_points, n * p * sizeof(Point), hipMemcpyHostToDevice));


        hipEvent_t start, stop;
        unsigned int bytes = n * (sizeof(Point));
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));
        float elapsed_time = 0;
        checkCudaErrors(hipEventRecord(start, 0));

        quickSelectAndPartition(d_points, n , p, 0);


        checkCudaErrors(hipEventRecord(stop, 0));
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        elapsed_time = elapsed_time ;
        double throughput = 1.0e-9 * ((double)bytes) / (elapsed_time * 1e-3);
        printf("quick_selection, Throughput = %.4f GB/s, Time = %.5f ms, Size = %d, p = %d, NumDevsUsed = %d\n",
               throughput, elapsed_time, n, p, 1);
        checkCudaErrors(
            hipMemcpy(h_points, d_points, n * p * sizeof(Point), hipMemcpyDeviceToHost));

        Point *t_points;
        int nn = n;
        for (int i = 0; i < p; ++i)
        {
            t_points = h_points + i * (1 + n);
            nn =  n - i;
            for (int i = 0; i < nn / 2; ++i)
            {
                ASSERT_LE(t_points[i].p[0], t_points[nn / 2].p[0]) << "Faild with n = " << nn << " and p " << p;
            }
            for (int i = n / 2; i < nn; ++i)
            {
                ASSERT_GE(t_points[i].p[0], t_points[nn / 2].p[0]) << "Faild with n = " << nn << " and p " << p;
            }
        }

        checkCudaErrors(
            hipFree(d_points));
        free(h_points);
        hipDeviceSynchronize();
        hipDeviceReset();
    }
}
