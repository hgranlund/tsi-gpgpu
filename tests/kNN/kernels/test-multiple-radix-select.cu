#include "multiple-radix-select.cuh"
#include <knn_gpgpu.h>
#include "test-common.cuh"

#define debug 0
#define THREADS_PER_BLOCK 1024U
#define MAX_BLOCK_DIM_SIZE 65535U


void printPoints2(struct Point *l, int n)
{
    int i;
    if (debug)
    {
        // printf("[(%3.1f, %3.1f, %3.1f)", l[0].p[0], l[0].p[1], l[0].p[2]);
        printf("[%3.1f, ", l[0].p[0]);
        for (i = 1; i < n; ++i)
        {
            printf(", %3.1f, ", l[i].p[0]);
            // printf(", (%3.1f, %3.1f, %3.1f)", l[i].p[0], l[i].p[1], l[i].p[2]);
        }
        printf("]\n");
    }
}


int cpu_partition(struct Point *data, int l, int u, int bit)
{
    unsigned int radix = (1 << (31 - bit));
    struct Point *temp = (struct Point *)malloc(((u - l) + 1) * sizeof(Point));
    int pos = 0;
    for (int i = l; i <= u; i++)
    {
        if (((*(int *) & (data[i].p[0]))&radix))
        {
            temp[pos++] = data[i];
        }
    }
    int result = u - pos;
    for (int i = l; i <= u; i++)
    {
        if (!((*(int *) & (data[i]))&radix))
        {
            temp[pos++] = data[i];
        }
    }
    pos = 0;
    for (int i = u; i >= l; i--)
    {
        data[i] = temp[pos++];
    }

    free(temp);
    return result;
}

struct Point cpu_radixselect(struct Point *data, int l, int u, int m, int bit)
{

    struct Point t;
    t.p[0] = 0;
    t.p[1] = 0;
    t.p[2] = 0;
    if (l == u) return (data[l]);
    if (bit > 32)
    {
        printf("cpu_radixselect fail!\n");
        return t;
    }
    int s = cpu_partition(data, l, u, bit);
    if (s >= m) return cpu_radixselect(data, l, s, m, bit + 1);
    return cpu_radixselect(data, s + 1, u, m, bit + 1);
}

TEST(multiple_radix_select, correctness)
{
    struct Point *h_points, *d_points, *d_swap;
    int n, p, *d_partition, *h_steps, *d_steps, dim = 0;
    for (n = 10; n <= 8000; n += 1000)
    {
        p = 2;
        h_steps = (int *) malloc(p * 2 * sizeof(int));
        h_steps[0] = 0;
        h_steps[1] = n / p;
        h_steps[2] = n / p + 1;
        h_steps[3] = n;

        h_points = (struct Point *) malloc(n * sizeof(Point));
        readPoints("../tests/data/10000_points.data", n, h_points);
        // readPoints("/home/simenhg/workspace/tsi-gpgpu/tests/data/100_mill_points.data", n, h_points);
        // populatePointSRosetta(h_points, n);
        // printPoints2(h_points, n / 2);


        checkCudaErrors(hipMalloc((void **)&d_points, n  * sizeof(Point)));
        checkCudaErrors(hipMalloc((void **)&d_swap, n  * sizeof(Point)));
        checkCudaErrors(hipMalloc((void **)&d_partition, n  * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_steps, p * 2 * sizeof(int)));

        checkCudaErrors(hipMemcpy(d_steps, h_steps, p * 2 * sizeof(int), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_points, h_points, n  * sizeof(Point), hipMemcpyHostToDevice));

        multiRadixSelectAndPartition(d_points, d_swap, d_partition, d_steps, n, p, dim);

        checkCudaErrors(hipMemcpy(h_points, d_points, n  * sizeof(Point), hipMemcpyDeviceToHost));
        // printPoints2(h_points, n / 2);

        ASSERT_TREE_LEVEL_OK(h_points, h_steps, n, p, dim);

        checkCudaErrors(hipFree(d_points));
        checkCudaErrors(hipFree(d_steps));
        checkCudaErrors(hipFree(d_swap));
        checkCudaErrors(hipFree(d_partition));
        free(h_points);
        free(h_steps);
        hipDeviceSynchronize();
        hipDeviceReset();
    }
}

TEST(multiple_radix_select, correctness_dim)
{
    struct Point *h_points, *d_points, *d_swap;
    int n = 1024,
        p = 2,
        dim = 0,
        *d_partition,
        *h_steps,
        *d_steps;

    h_steps = (int *) malloc(p * 2 * sizeof(int));
    h_steps[0] = 0;
    h_steps[1] = n / p;
    h_steps[2] = n / p + 1;
    h_steps[3] = n;

    h_points = (struct Point *) malloc(n * sizeof(Point));
    readPoints("../tests/data/10000_points.data", n, h_points);

    checkCudaErrors(hipMalloc((void **)&d_points, n  * sizeof(Point)));
    checkCudaErrors(hipMalloc((void **)&d_swap, n  * sizeof(Point)));
    checkCudaErrors(hipMalloc((void **)&d_partition, n  * sizeof(int)));
    checkCudaErrors(hipMalloc((void **)&d_steps, p * 2 * sizeof(int)));

    checkCudaErrors(hipMemcpy(d_steps, h_steps, p * 2 * sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_points, h_points, n  * sizeof(Point), hipMemcpyHostToDevice));

    for (dim = 0; dim < 3; dim++)
    {
        multiRadixSelectAndPartition(d_points, d_swap, d_partition, d_steps, n, p, dim);
        checkCudaErrors(hipMemcpy(h_points, d_points, n  * sizeof(Point), hipMemcpyDeviceToHost));

        ASSERT_TREE_LEVEL_OK(h_points, h_steps, n, p, dim);

    }
    checkCudaErrors(hipFree(d_points));
    checkCudaErrors(hipFree(d_steps));
    checkCudaErrors(hipFree(d_swap));
    checkCudaErrors(hipFree(d_partition));
    free(h_points);
    free(h_steps);
    hipDeviceSynchronize();
}

TEST(multiple_radix_select, timing)
{
    struct Point *h_points, *d_points, *d_swap;
    int n, p, *d_partition, *h_steps, *d_steps;
    for (n = 8388608; n <= 8388608; n <<= 1)
    {
        p = 2;
        h_steps = (int *) malloc(p * 2 * sizeof(int));
        h_steps[0] = 0;
        h_steps[1] = n / p;
        h_steps[2] = n / p + 1;
        h_steps[3] = n;

        h_points = (struct Point *) malloc(n * sizeof(Point));
        populatePointSs(h_points, n);

        checkCudaErrors(hipMalloc((void **)&d_points, n  * sizeof(Point)));
        checkCudaErrors(hipMalloc((void **)&d_swap, n  * sizeof(Point)));
        checkCudaErrors(hipMalloc((void **)&d_partition, n  * sizeof(int)));
        checkCudaErrors(hipMalloc((void **)&d_steps, p * 2 * sizeof(int)));

        checkCudaErrors(hipMemcpy(d_steps, h_steps, p * 2 * sizeof(int), hipMemcpyHostToDevice));


        float elapsed_time = 0;
        hipEvent_t start, stop;

        cudaStartTiming(start, stop, elapsed_time);

        checkCudaErrors(hipMemcpy(d_points, h_points, n  * sizeof(Point), hipMemcpyHostToDevice));
        multiRadixSelectAndPartition(d_points, d_swap, d_partition, d_steps, n, p, 0);

        cudaStopTiming(start, stop, elapsed_time);

        int bytes = n * (sizeof(float)) ;
        printCudaTiming(elapsed_time, bytes, n);

        checkCudaErrors(hipFree(d_points));
        checkCudaErrors(hipFree(d_steps));
        checkCudaErrors(hipFree(d_swap));
        checkCudaErrors(hipFree(d_partition));
        free(h_points);
        free(h_steps);
        hipDeviceReset();
    }
}
