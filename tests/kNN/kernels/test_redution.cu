#include "hip/hip_runtime.h"
// Includes
// #include <reduction.cuh>
#include <reduction-mod.cuh>
#include <knn_gpgpu.h>
#include "test-common.cuh"

#define inf 0x7f800000
#define debug 0
float cpu_min(float *in, int num_els)
{
    float min = inf;

    for (int i = 0; i < num_els; i++)
        min = in[i] < min ? in[i] : min;

    return min;
}

void printDistArray(Distance *l, int n)
{
    int i;
    if (debug)
    {
        printf("[(%d - %3.1f)", l[0].index, l[0].value );
        for (i = 1; i < n; ++i)
        {
            printf(", (%d - %3.1f)", l[i].index, l[i].value );
        }
        printf("]\n");
    }
}

void printIntArray(int *l, int n)
{
    int i;
    if (debug)
    {
        printf("[%4d", l[0] );
        for (i = 1; i < n; ++i)
        {
            printf(", %4d", l[i] );
        }
        printf("]\n");
    }
}


// TEST(min_reduce, min_reduce){

//   Distance *h_dist;
//   int i,n;
//   for (n = 11; n <=11; n +=2)
//   {

//     h_dist = (Distance*) malloc(n*sizeof(Distance));

//     srand ( (unsigned int)time(NULL) );
//     for (i=0 ; i<n; i++)
//     {
//       h_dist[i].value    = n-i-1;
//       h_dist[i].value=i;
//     }
//     // printf("########\n");
//     // printFloatArray(list,n);
//     // printIntArray(ind_1,n);

//     Distance *d_dist;

//     hipMalloc( (void **) &d_dist, n* sizeof(Distance));

//     hipMemcpy(d_dist,h_dist, n*sizeof(Distance), hipMemcpyHostToDevice);

//     knn_min_reduce(d_dist, n);

//     hipMemcpy(h_dist,d_dist, n*sizeof(Distance), hipMemcpyDeviceToHost);

//     ASSERT_LE(h_dist[0].value, 0)  << "Faild with n = " << n;
//     ASSERT_LE(h_dist[0].index, n-1)  << "Faild with n = " << n;

//     hipFree(d_dist);
//     free(h_dist);
//   }
// }

TEST(min_reduce, correcness)
{
    hipDeviceReset();

    Distance *h_dist;
    unsigned int i, n;
    for (n = 2; n <= 30000000; n <<= 1)
    {

        h_dist = (Distance *) malloc(n * sizeof(Distance));

        srand ( (unsigned int)time(NULL) );
        for (i = 0 ; i < n; i++)
        {
            h_dist[i].value    = (float) n - i - 1;
            h_dist[i].index = i;
        }
        // printf("########\n");
        // printDistArray(h_dist,n);
        // printIntArray(ind_1,n);

        Distance *d_dist;

        hipMalloc( (void **) &d_dist, n * sizeof(Distance));

        hipMemcpy(d_dist, h_dist, n * sizeof(Distance), hipMemcpyHostToDevice);

        dist_min_reduce(d_dist, n);

        hipMemcpy(h_dist, d_dist, n * sizeof(Distance), hipMemcpyDeviceToHost);

        // printDistArray(h_dist,n);

        ASSERT_EQ(h_dist[0].value, 0)  << "Faild with n = " << n;
        ASSERT_EQ(h_dist[0].index, n - 1)  << "Faild with n = " << n;
        hipFree(d_dist);
        free(h_dist);
        hipDeviceSynchronize();
        hipDeviceReset();
    }
}

TEST(min_reduce, timing)
{
    hipDeviceSynchronize();
    hipDeviceReset();
    Distance *h_dist;
    Distance *d_dist;
    unsigned int i, n;
    n = 8388608;
    h_dist = (Distance *) malloc(n * sizeof(Distance));

    srand ( (unsigned int)time(NULL) );
    for (i = 0 ; i < n; i++)
    {
        h_dist[i].value    = (float)n - i - 1;
        h_dist[i].index = i;
    }

    hipMalloc( (void **) &d_dist, n * sizeof(Distance));
    hipMemcpy(d_dist, h_dist, n * sizeof(Distance), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    unsigned int bytes = n * (sizeof(Distance) + sizeof(int));
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float elapsed_time = 0;

    checkCudaErrors(hipEventRecord(start, 0));


    dist_min_reduce(d_dist, n);

    checkCudaErrors(hipEventRecord(stop, 0));
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    elapsed_time = elapsed_time ;
    double throughput = 1.0e-9 * ((double)bytes) / (elapsed_time * 1e-3);
    printf("Reduction_mod, Throughput = %.4f GB/s, Time = %.5f ms, Size = %u Elements, NumDevsUsed = %d\n",
           throughput, elapsed_time, n, 1);

    hipMemcpy(h_dist, d_dist, n * sizeof(Distance), hipMemcpyDeviceToHost);

    ASSERT_LE(h_dist[0].value, 0)  << "Faild with n = " << n;
    ASSERT_LE(h_dist[0].index, n - 1)  << "Faild with n = " << n;

    hipFree(d_dist);
    free(h_dist);
    hipDeviceSynchronize();
    hipDeviceReset();
}
