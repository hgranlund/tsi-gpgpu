#include "hip/hip_runtime.h"
// Includes
#include <reduction.cuh>
#include <knn_gpgpu.h>
#include <stdio.h>
#include <gtest/gtest.h>

#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <assert.h>




#define inf 0x7f800000
#define debug 1
float cpu_min(float* in, int num_els)
{
  float min = inf;

  for(int i = 0; i < num_els; i++)
    min = in[i] < min ? in[i] : min;

  return min;
}

void printFloatArray(float* l, int n){
  int i;
  if (debug)
  {
    printf("[%3.1f", l[0] );
      for (i = 1; i < n; ++i)
      {
        printf(", %3.1f", l[i] );
      }
      printf("]\n");
    }
  }

  void printIntArray(int* l, int n){
    int i;
    if (debug)
    {
      printf("[%4d", l[0] );
        for (i = 1; i < n; ++i)
        {
          printf(", %4d", l[i] );
        }
        printf("]\n");
      }
    }


    TEST(kernels, min_reduce){

      float *h_list;
      int *h_ind;
      int i,n;
      for (n = 11; n <=11; n +=2)
      {

        h_list = (float*) malloc(n*sizeof(float));
        h_ind = (int*) malloc(n*sizeof(int));
        srand(time(NULL));
        for (i=0 ; i<n; i++)
        {
          h_list[i]    = n-i-1;
          h_ind[i]=i;
        }
        // printf("########\n");
        // printFloatArray(list,n);
        // printIntArray(ind_1,n);

        float *d_list;
        int *d_ind;

        hipMalloc( (void **) &d_list, n* sizeof(float));
        hipMalloc( (void **) &d_ind, n* sizeof(int));

        hipMemcpy(d_list,h_list, n*sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_ind,h_ind, n*sizeof(int), hipMemcpyHostToDevice);

        knn_min_reduce(d_list, d_ind, n);

        hipMemcpy(h_list,d_list, n*sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_ind,d_ind, n*sizeof(int), hipMemcpyDeviceToHost);

        ASSERT_LE(h_list[0], 0)  << "Faild with n = " << n;
        ASSERT_LE(h_ind[0], n-1)  << "Faild with n = " << n;

        hipFree(d_list);
        hipFree(d_ind);
        free(h_list);
        free(h_ind);
      }
    }

