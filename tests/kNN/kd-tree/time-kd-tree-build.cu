#include "knn_gpgpu.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

void writePoints(char *file_path, int n, struct Point *points)
{
    printf("writing points...\n");

    FILE *file = fopen(file_path, "w");
    if (file == NULL)
    {
        fputs ("File error\n", stderr);
        exit (1);
    }
    for (int i = 0; i < n; ++i)
    {
        fwrite(&points[i].p, sizeof(float), 3, file);
    }
    fclose(file);
}

void readPoints(const char *file_path, int n, struct Point *points)
{
    printf("Reading points...\n");

    FILE *file = fopen(file_path, "rb");
    if (file == NULL)
    {
        fputs ("File error\n", stderr);
        exit (1);
    }
    for (int i = 0; i < n; ++i)
    {
        fread(&points[i].p, sizeof(float), 3, file);
    }

    fclose(file);
}

void populatePoints(struct Point *points, int n)
{
    int i;
    srand(time(NULL));

    for (i = 0; i < n; ++i)
    {
        struct Point t;
        t.p[0] = rand(), t.p[1] = rand(), t.p[2] = rand();
        points[i] = t;
    }
}

int main(int argc, char const *argv[])
{
    int n, nu, ni = 8388608,
               step = 250000;
    bool from_file = 0;
    n = nu = ni;

    if (argc == 2)
    {
        nu = ni = atoi(argv[1]);
        printf("Running kd-tree-build with n = %d\n", nu);
    }
    else if (argc == 3)
    {
        nu = ni = atoi(argv[1]);
        from_file = 1;
        printf("Running kd-tree-build from file '%s' with n = %d\n", argv[2], nu);
    }
    else if (argc == 4)
    {
        nu = atoi(argv[1]);
        ni = atoi(argv[2]);
        step = atoi(argv[3]);
        printf("Running kd-tree-build from n = %d to n = %d with step = %d\n", nu, ni, step);
    }
    else
    {
        printf("Running kd-tree-build with n = %d\n", nu);
    }

    for (n = nu; n <= ni ; n += step)
    {
        struct Node *points_out = (struct Node *) malloc(n  * sizeof(Node));
        struct Point *points = (struct Point *) malloc(n  * sizeof(Point));

        if (from_file)
        {
            readPoints(argv[2], n, points);
        }
        else
        {
            populatePoints(points, n);
        }

        hipEvent_t start, stop;
        float elapsed_time = 0;
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));
        checkCudaErrors(hipEventRecord(start, 0));

        buildKdTree(points, n, points_out);

        checkCudaErrors(hipEventRecord(stop, 0));
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);

        printf("buildKdTree_naive,  Time = %.5f ms, Size = %u Elements, NumDevsUsed = %d\n",
               elapsed_time, n, 1);

        free(points);
        free(points_out);
        hipDeviceReset();
    }
    return 0;
}

