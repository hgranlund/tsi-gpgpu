#include "hip/hip_runtime.h"
#include "kd-tree-naive.cuh"
#include "knn_gpgpu.h"
#include "point.h"

#include "stdio.h"
#include "hip/hip_runtime_api.h"
#include "gtest/gtest.h"

#define debug 0

void writePoints(char *file_path, int n, PointS *points)
{
    printf("writing points...\n");

    FILE *file = fopen(file_path, "w");
    if (file == NULL)
    {
        fputs ("File error\n", stderr);
        exit (1);
    }
    for (int i = 0; i < n; ++i)
    {
        fwrite(&points[i].p, sizeof(float), 3, file);
    }
    fclose(file);
}


void readPoints(const char *file_path, int n, PointS *points)
{
    printf("Reading points...\n");

    FILE *file = fopen(file_path, "rb");
    if (file == NULL)
    {
        fputs ("File error\n", stderr);
        exit (1);
    }
    for (int i = 0; i < n; ++i)
    {
        fread(&points[i].p, sizeof(float), 3, file);
    }

    fclose(file);
}


__host__  void h_printPointsArray__(PointS *l, int n, char *s, int l_debug = 0)
{
    if (debug || l_debug)
    {
        printf("%10s: [ ", s);
        for (int i = 0; i < n; ++i)
        {
            printf("%3.1f, ", l[i].p[0]);
        }
        printf("]\n");
    }
}


int h_index(int i, int j, int n)
{
    return i + j * n;
}

void h_swap(PointS *points, int a, int b, int n)
{
    PointS t = points[a];
    points[a] = points[b], points[b] = t;
}

int midpoint(int lower, int upper)
{
    return (int) floor((float)(upper - lower) / 2) + lower;
}

void print_tree(PointS *tree, int level, int lower, int upper, int n)
{
    if (debug)
    {
        if (lower >= upper)
        {
            return;
        }

        int i, r = midpoint(lower, upper);

        printf("|");
        for (i = 0; i < level; ++i)
        {
            printf("--");
        }
        printf("(%3.1f, %3.1f, %3.1f)\n", tree[r].p[0], tree[r].p[1], tree[r].p[2]);

        print_tree(tree, 1 + level, lower, r, n);
        print_tree(tree, 1 + level, r + 1, upper, n);
    }
}

void populatePoints(PointS *points, int n)
{
    srand(time(NULL));
    for (int i = 0; i < n; ++i)
    {
        PointS t;
        t.p[0] = rand();
        t.p[1] = rand();
        t.p[2] = rand();
        points[i]    = t;
    }
}

int main(int argc, char const *argv[])
{
    int n, nu, ni = 8388608,
               step = 250000;
    bool from_file = 0;
    n = nu = ni;
    if (argc == 2)
    {
        nu = ni = atoi(argv[1]);
        printf("Running kd-tree-build with n = %d\n", nu);
    }
    else if (argc == 3)
    {
        nu = ni = atoi(argv[1]);
        from_file = 1;
        printf("Running kd-tree-build from file '%s' with n = %d\n", argv[2], nu);
    }
    else if (argc == 4)
    {
        nu = atoi(argv[1]);
        ni = atoi(argv[2]);
        step = atoi(argv[3]);
        printf("Running kd-tree-build from n = %d to n = %d with step = %d\n", nu, ni, step);
    }
    else
    {
        printf("Running kd-tree-build with n = %d\n", nu);
    }

    for (n = nu; n <= ni ; n += step)
    {
        hipDeviceReset();
        PointS *points;
        Point *points_out;
        points_out = (Point *) malloc(n  * sizeof(Point));
        points = (PointS *) malloc(n  * sizeof(PointS));

        if (from_file)
        {
            readPoints(argv[2], n, points);
        }
        else
        {
            populatePoints(points, n);
        }
        hipEvent_t start, stop;
        unsigned int bytes = n * (sizeof(PointS));
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));
        float elapsed_time = 0;

        checkCudaErrors(hipEventRecord(start, 0));

        build_kd_tree(points, n, points_out);

        checkCudaErrors(hipEventRecord(stop, 0));
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        elapsed_time = elapsed_time ;
        double throughput = 1.0e-9 * ((double)bytes) / (elapsed_time * 1e-3);
        printf("build_kd_tree_naive, Throughput = %.4f GB/s, Time = %.5f ms, Size = %u Elements, NumDevsUsed = %d\n",
               throughput, elapsed_time, n, 1);
        free(points);
        free(points_out);
    }
    return 0;

}

