#include "knn_gpgpu.h"
#include "point.h"
#include "stdio.h"
#include "hip/hip_runtime_api.h"
#include "gtest/gtest.h"

#define debug 0

void writePoints(char *file_path, int n, PointS *points)
{
    printf("writing points...\n");

    FILE *file = fopen(file_path, "w");
    if (file == NULL)
    {
        fputs ("File error\n", stderr);
        exit (1);
    }
    for (int i = 0; i < n; ++i)
    {
        fwrite(&points[i].p, sizeof(float), 3, file);
    }
    fclose(file);
}


void readPoints(const char *file_path, int n, PointS *points)
{
    printf("Reading points...\n");

    FILE *file = fopen(file_path, "rb");
    if (file == NULL)
    {
        fputs ("File error\n", stderr);
        exit (1);
    }
    for (int i = 0; i < n; ++i)
    {
        fread(&points[i].p, sizeof(float), 3, file);
    }

    fclose(file);
}

void populatePoints(PointS *points, int n)
{
    srand(time(NULL));
    for (int i = 0; i < n; ++i)
    {
        PointS t;
        t.p[0] = rand();
        t.p[1] = rand();
        t.p[2] = rand();
        points[i]    = t;
    }
}

int main(int argc, char const *argv[])
{
    int n, nu, ni = 1024,
               step = 250000,
               k = 1;
    bool from_file = 0;
    n = nu = ni;
    if (argc == 2)
    {
        nu = ni = atoi(argv[1]);
        printf("Running kd-search-all with n = %d\n", nu);
    }
    else if (argc == 3)
    {
        nu = ni = atoi(argv[1]);
        from_file = 1;
        printf("Running kd-search-all from file '%s' with n = %d\n", argv[2], nu);
    }
    else if (argc == 4)
    {
        nu = atoi(argv[1]);
        ni = atoi(argv[2]);
        step = atoi(argv[3]);
        printf("Running kd-search-all from n = %d to n = %d with step = %d\n", nu, ni, step);
    }
    else
    {
        printf("Running kd-search-all with n = %d\n", nu);
    }

    for (n = nu; n <= ni ; n += step)
    {
        hipDeviceReset();
        PointS *points;
        Point *points_out;
        points_out = (Point *) malloc(n  * sizeof(Point));
        points = (PointS *) malloc(n  * sizeof(PointS));
        int *result = (int *) malloc(n * k * sizeof(int));

        if (from_file)
        {
            readPoints(argv[2], n, points);
        }
        else
        {
            populatePoints(points, n);
        }
        hipEvent_t start, stop;
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));
        float elapsed_time_build = 0;

        checkCudaErrors(hipEventRecord(start, 0));

        build_kd_tree(points, n, points_out);

        checkCudaErrors(hipEventRecord(stop, 0));
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time_build, start, stop);


        hipDeviceReset();
        checkCudaErrors(hipEventCreate(&start));
        checkCudaErrors(hipEventCreate(&stop));
        float elapsed_time_search = 0;

        checkCudaErrors(hipEventRecord(start, 0));

        queryAll(points_out, points_out, n, n, k, result);

        checkCudaErrors(hipEventRecord(stop, 0));
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time_search, start, stop);

        printf("kd-search-all,  Build Time = %.5f ms, Query Time = %.5f ms, Total time = %.5f ms, Size = %u Elements, NumDevsUsed = %d\n",
               elapsed_time_build, elapsed_time_search, elapsed_time_build + elapsed_time_search, n, 1);

        free(points);
        free(result);
        free(points_out);
    }
    return 0;

}

