#include <knn_gpgpu.h>
#include <point.h>
#include "test-common.cuh"

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "gtest/gtest.h"

TEST(kd_tree_build, correctness)
{
    int i, n = 8;
    float temp;

    struct PointS *points = (PointS *) malloc(n  * sizeof(PointS));
    struct Point *points_out = (Point *) malloc(n  * sizeof(Point));
    struct Point *expected_points = (Point *) malloc(n * sizeof(Point));

    srand(time(NULL));

    for (i = 0; i < n; ++i)
    {
        Point t2;
        PointS t;
        temp = n - i - 1;

        t.p[0] = temp, t.p[1] = temp, t.p[2] = temp;
        t2.p[0] = i, t2.p[1] = i, t2.p[2] = i;

        points[i] = t;
        expected_points[i] = t2;
    }

    build_kd_tree(points, n , points_out);

    ASSERT_TREE_EQ(points_out, expected_points, n);

    free(points);
    free(points_out);
    free(expected_points);
}

TEST(kd_tree_build, timing)
{
    int n;

    // for (n = 8388608; n <= 8388608 ; n += 250000)
    for (n = 1024; n <= 1024 ; n += 250000)
    {
        struct PointS *points = (PointS *) malloc(n * sizeof(PointS));
        struct Point *points_out = (Point *) malloc(n * sizeof(Point));

        populatePointSs(points, n);
        populatePoints(points_out, n);

        float elapsed_time;
        int bytes = n * (sizeof(PointS));
        hipEvent_t start, stop;

        cudaStartTiming(start, stop, elapsed_time);
        build_kd_tree(points, n, points_out);
        cudaStopTiming(start, stop, elapsed_time);
        printCudaTiming(elapsed_time, bytes, n);

        free(points);
        free(points_out);
        hipDeviceReset();
    }
}

TEST(kd_tree_build, wikipedia_example)
{
    hipDeviceReset();
    int n = 6;
    struct PointS *points = (PointS *) malloc(n  * sizeof(PointS));
    struct Point *points_out = (Point *) malloc(n  * sizeof(Point));
    struct PointS *points_correct = (PointS *) malloc(n  * sizeof(PointS));

    points[0].p[0] = 2, points[0].p[1] = 3, points[0].p[2] = 0;
    points[1].p[0] = 5, points[1].p[1] = 4, points[1].p[2] = 0;
    points[2].p[0] = 9, points[2].p[1] = 6, points[2].p[2] = 0;
    points[3].p[0] = 4, points[3].p[1] = 7, points[3].p[2] = 0;
    points[4].p[0] = 8, points[4].p[1] = 1, points[4].p[2] = 0;
    points[5].p[0] = 7, points[5].p[1] = 2, points[5].p[2] = 0;

    build_kd_tree(points, n, points_out);

    points_correct[0].p[0] = 2, points_correct[0].p[1] = 3, points_correct[0].p[2] = 0;
    points_correct[1].p[0] = 5, points_correct[1].p[1] = 4, points_correct[1].p[2] = 0;
    points_correct[2].p[0] = 4, points_correct[2].p[1] = 7, points_correct[2].p[2] = 0;
    points_correct[3].p[0] = 7, points_correct[3].p[1] = 2, points_correct[3].p[2] = 0;
    points_correct[4].p[0] = 8, points_correct[4].p[1] = 1, points_correct[4].p[2] = 0;
    points_correct[5].p[0] = 9, points_correct[5].p[1] = 6, points_correct[5].p[2] = 0;

    for (int i = 0; i < n; ++i)
    {
        ASSERT_EQ(points_correct[i].p[0], points_out[i].p[0]) << "failed at i = " << i;
        ASSERT_EQ(points_correct[i].p[1], points_out[i].p[1]) << "failed at i = " << i;
        ASSERT_EQ(points_correct[i].p[2], points_out[i].p[2]) << "failed at i = " << i;
    }
    free(points_out);
    free(points);
    free(points_correct);
}
