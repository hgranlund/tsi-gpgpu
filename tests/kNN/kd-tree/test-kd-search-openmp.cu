#include <float.h>
#include <math.h>

#include "test-common.cuh"
#include "kd-search-openmp.cuh"
#include <knn_gpgpu.h>

bool isExpectedPoint_(struct Node *tree, int n, int k,  float qx, float qy, float qz, float ex, float ey, float ez)
{
    struct Point query_point;

    struct SPoint *s_stack_ptr = (struct SPoint *)malloc(51 * sizeof(struct SPoint));
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc((k + 1) * sizeof(struct KPoint));

    int *result = (int *) malloc(k * sizeof(int));

    query_point.p[0] = qx, query_point.p[1] = qy, query_point.p[2] = qz;

    kNN(query_point, tree, n, k, result, s_stack_ptr, k_stack_ptr);

    float actual = tree[result[0]].p[0] + tree[result[0]].p[1] + tree[result[0]].p[2];
    float expected = ex + ey + ez;

    // printf(">> WP tree\nsearching for (%3.1f, %3.1f, %3.1f)\n"
    //        "found (%3.1f, %3.1f, %3.1f) seen %d nodes\n\n",
    //        qx, qy, qz,
    //        tree[result[0]].p[0], tree[result[0]].p[1], tree[result[0]].p[2], visited);

    free(s_stack_ptr);
    free(k_stack_ptr);
    free(result);

    if (actual == expected)
    {
        return true;
    }
    return false;
}

TEST(kd_search_openmp, isEmpty)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(4 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1;

    initStack(&stack);

    value1.index = 10;

    ASSERT_TRUE(isEmpty(stack));

    stack[0] = value1;
    stack++;
    ASSERT_FALSE(isEmpty(stack));

    free(stack_ptr);
}

TEST(kd_search_openmp, push)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(3 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1,
                    value2;

    initStack(&stack);

    value1.index = 1;
    value2.index = 3;

    push(&stack, value1);
    push(&stack, value2);
    ASSERT_EQ(value1.index, stack_ptr[1].index);
    ASSERT_EQ(value2.index, stack_ptr[2].index);

    free(stack_ptr);
}

TEST(kd_search_openmp, pop)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(4 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1,
                    value2,
                    value3;

    initStack(&stack);

    value1.index = 1;
    value2.index = 2;
    value3.index = 3;

    stack[0] = value1;
    stack[1] = value2;
    stack[2] = value3;
    stack += 3;

    ASSERT_EQ(value3.index, pop(&stack).index);
    ASSERT_EQ(value2.index, pop(&stack).index);
    ASSERT_EQ(value1.index, pop(&stack).index);

    free(stack_ptr);
}

TEST(kd_search_openmp, peek)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(4 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1;

    initStack(&stack);

    value1.index = 10;

    ASSERT_EQ(-1, peek(stack).index);
    ASSERT_EQ(-1, peek(stack).index);

    push(&stack, value1);

    ASSERT_EQ(value1.index, peek(stack).index);
    ASSERT_EQ(value1.index, peek(stack).index);

    free(stack_ptr);
}

TEST(kd_search_openmp, initKStack)
{
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc(51 * sizeof(struct KPoint)),
                   *k_stack = k_stack_ptr;

    initKStack(&k_stack, 50);

    ASSERT_EQ(-1, k_stack[-1].dist);
    ASSERT_EQ(FLT_MAX, k_stack[0].dist);
    ASSERT_EQ(FLT_MAX, k_stack[49].dist);

    free(k_stack_ptr);
}

TEST(kd_search_openmp, insert)
{
    int n = 3;
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc(51 * sizeof(struct KPoint)),
                   *k_stack = k_stack_ptr;

    initKStack(&k_stack, n);
    struct KPoint a, b, c, d;

    a.dist = 1;
    b.dist = 2;
    c.dist = 3;
    d.dist = 0;

    insert(k_stack, a, n);
    ASSERT_EQ(FLT_MAX, look(k_stack, n).dist);
    ASSERT_EQ(a.dist, k_stack[0].dist);

    insert(k_stack, b, n);
    ASSERT_EQ(FLT_MAX, look(k_stack, n).dist);
    ASSERT_EQ(b.dist, k_stack[1].dist);

    insert(k_stack, c, n);
    ASSERT_EQ(c.dist, look(k_stack, n).dist);
    ASSERT_EQ(c.dist, k_stack[2].dist);

    insert(k_stack, d, n);
    ASSERT_EQ(b.dist, look(k_stack, n).dist);
    ASSERT_EQ(d.dist, k_stack[0].dist);

    free(k_stack_ptr);
}

TEST(kd_search_openmp, insert_k_is_one)
{
    int n = 1;
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc(51 * sizeof(struct KPoint)),
                   *k_stack = k_stack_ptr;

    initKStack(&k_stack, n);
    struct KPoint a, b;

    a.dist = 1;
    b.dist = 0;

    insert(k_stack, a, n);
    ASSERT_EQ(a.dist, look(k_stack, n).dist);
    ASSERT_EQ(a.dist, k_stack[0].dist);

    insert(k_stack, b, n);
    ASSERT_EQ(b.dist, look(k_stack, n).dist);
    ASSERT_EQ(b.dist, k_stack[0].dist);

    free(k_stack_ptr);
}

TEST(kd_search_openmp, upDim)
{
    int dim = 0;

    upDim(&dim);
    ASSERT_EQ(1, dim);

    upDim(&dim);
    ASSERT_EQ(2, dim);

    upDim(&dim);
    ASSERT_EQ(0, dim);

    upDim(&dim);
    ASSERT_EQ(1, dim);
}

TEST(kd_search_openmp, correctness_with_k)
{
    int n = 6,
        k = 3;

    int *result = (int *) malloc(k * sizeof(int));

    struct Point *points = (struct Point *) malloc(n  * sizeof(struct Point));
    struct Node *tree = (struct Node *) malloc(n  * sizeof(struct Node));

    struct SPoint *s_stack_ptr = (struct SPoint *)malloc(51 * sizeof(struct SPoint));
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc((k + 1) * sizeof(struct KPoint));

    points[0].p[0] = 2, points[0].p[1] = 3, points[0].p[2] = 0;
    points[1].p[0] = 5, points[1].p[1] = 4, points[1].p[2] = 0;
    points[2].p[0] = 9, points[2].p[1] = 6, points[2].p[2] = 0;
    points[3].p[0] = 4, points[3].p[1] = 7, points[3].p[2] = 0;
    points[4].p[0] = 8, points[4].p[1] = 1, points[4].p[2] = 0;
    points[5].p[0] = 7, points[5].p[1] = 2, points[5].p[2] = 0;

    hipDeviceReset();
    build_kd_tree(points, n, tree);

    hipDeviceReset();
    kNN(points[4], tree, n, k, result, s_stack_ptr, k_stack_ptr);

    ASSERT_EQ(4, result[0]);
    ASSERT_EQ(3, result[1]);
    ASSERT_EQ(1, result[2]);

    free(points);
    free(tree);

    free(s_stack_ptr);
    free(k_stack_ptr);
}

TEST(kd_search_openmp, correctness_with_10000_points_file)
{
    int n, k = 1;

    for (n = 1000; n <= 10000; n += 1000)
    {
        struct Point *points = (struct Point *) malloc(n  * sizeof(struct Point));
        struct Node *tree = (struct Node *) malloc(n  * sizeof(struct Node));

        srand(time(NULL));

        readPoints("../tests/data/10000_points.data", n, points);

        hipDeviceReset();
        build_kd_tree(points, n, tree);

        // printTree(tree, 0, n / 2);

        int *result = (int *) malloc(k * sizeof(int));

        int i,
            test_runs = n;

        struct SPoint *stack_ptr = (struct SPoint *)malloc(51 * sizeof(struct SPoint));
        struct KPoint *k_stack_ptr = (struct KPoint *) malloc((k + 1) * sizeof(struct KPoint));

        for (i = 0; i < test_runs; ++i)
        {
            hipDeviceReset();
            kNN(points[i], tree, n, k, result, stack_ptr, k_stack_ptr);

            // printf("Looking for (%3.1f, %3.1f, %3.1f), found (%3.1f, %3.1f, %3.1f)\n",
            //        tree[i].p[0], tree[i].p[1], tree[i].p[2],
            //        tree[result[0]].p[0], tree[result[0]].p[1], tree[result[0]].p[2]);

            ASSERT_EQ(points[i].p[0], tree[result[0]].p[0]) << "Failed at i = " << i << " with n = " << n ;
            ASSERT_EQ(points[i].p[1], tree[result[0]].p[1]) << "Failed at i = " << i << " with n = " << n;
            ASSERT_EQ(points[i].p[2], tree[result[0]].p[2]) << "Failed at i = " << i << " with n = " << n;
        }

        free(tree);
        free(result);
        free(points);
        free(stack_ptr);
        free(k_stack_ptr);
    };
};

// TEST(kd_search_openmp, queryAll_correctness_with_10000_points_file)
// {
//     int n, i, k = 1;

//     for (n = 1000; n <= 10000; n += 1000)
//     {
//         struct Point *points = (struct Point *) malloc(n  * sizeof(struct Point));
//         struct Node *tree = (struct Node *) malloc(n  * sizeof(struct Node));

//         srand(time(NULL));

//         readPoints("../tests/data/10000_points.data", n, points);

//         hipDeviceReset();
//         build_kd_tree(points, n, tree);

//         // printTree(tree, 0, n / 2);

//         int *result = (int *) malloc(n * k * sizeof(int));


//         queryAll(points, tree, n, n, 1, result);
//         for (i = 0; i < n; ++i)
//         {
//             ASSERT_EQ(points[i].p[0], tree[result[i]].p[0]) << "Failed at i = " << i << " with n = " << n ;
//             ASSERT_EQ(points[i].p[1], tree[result[i]].p[1]) << "Failed at i = " << i << " with n = " << n;
//             ASSERT_EQ(points[i].p[2], tree[result[i]].p[2]) << "Failed at i = " << i << " with n = " << n;
//         }

//         free(tree);
//         free(result);
//         free(points);
//     };
// };

TEST(kd_search_openmp, knn_wikipedia_example)
{
    int n = 6,
        k = 1;

    struct Point *points = (struct Point *) malloc(n  * sizeof(struct Point));
    struct Node *tree = (struct Node *) malloc(n  * sizeof(struct Node));

    points[0].p[0] = 2, points[0].p[1] = 3, points[0].p[2] = 0;
    points[1].p[0] = 5, points[1].p[1] = 4, points[1].p[2] = 0;
    points[2].p[0] = 9, points[2].p[1] = 6, points[2].p[2] = 0;
    points[3].p[0] = 4, points[3].p[1] = 7, points[3].p[2] = 0;
    points[4].p[0] = 8, points[4].p[1] = 1, points[4].p[2] = 0;
    points[5].p[0] = 7, points[5].p[1] = 2, points[5].p[2] = 0;

    hipDeviceReset();
    build_kd_tree(points, n, tree);


    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 2, 3, 0, 2, 3, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 5, 4, 0, 5, 4, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 9, 6, 0, 9, 6, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 4, 7, 0, 4, 7, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 8, 1, 0, 8, 1, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 7, 2, 0, 7, 2, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 10, 10, 0, 9, 6, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 0, 0, 0, 2, 3, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 4, 4, 0, 5, 4, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 3, 2, 0, 2, 3, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 2, 6, 0, 4, 7, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 10, 0, 0, 8, 1, 0));
    ASSERT_EQ(true, isExpectedPoint_(tree, n, k, 0, 10, 0, 4, 7, 0));

    free(points);
    free(tree);
}

TEST(kd_search_openmp, query_all_wikipedia_example)
{
    int n = 6, k = 1;
    struct Point *points = (struct Point *) malloc(n * sizeof(struct Point));
    struct Node *tree = (struct Node *) malloc(n * sizeof(struct Node));
    int *result = (int *) malloc(n * k * sizeof(int));

    points[0].p[0] = 2, points[0].p[1] = 3, points[0].p[2] = 0;
    points[1].p[0] = 5, points[1].p[1] = 4, points[1].p[2] = 0;
    points[5].p[0] = 7, points[5].p[1] = 2, points[5].p[2] = 0;

    hipDeviceReset();
    build_kd_tree(points, n, tree);

    mpQueryAll(points, tree, n, n, 1, result);

    ASSERT_EQ(result[0], 0);
    ASSERT_EQ(result[1], 1);
    ASSERT_EQ(result[2], 5);
    ASSERT_EQ(result[3], 2);
    ASSERT_EQ(result[4], 4);
    ASSERT_EQ(result[5], 3);

    free(points);
    free(tree);
    free(result);
}

TEST(kd_search_openmp, knn_timing)
{
    int n, k = 1;

    for (n = 10000; n <= 10000; n += 1000)
    {
        struct Point *points = (struct Point *) malloc(n  * sizeof(struct Point));
        struct Node *tree = (struct Node *) malloc(n  * sizeof(struct Node));

        struct SPoint *stack_ptr = (struct SPoint *)malloc(51 * sizeof(struct SPoint));
        struct KPoint *k_stack_ptr = (struct KPoint *) malloc((k + 1) * sizeof(struct KPoint));

        int *result = (int *) malloc(k * sizeof(int));

        srand(time(NULL));

        readPoints("../tests/data/10000_points.data", n, points);

        hipDeviceReset();
        build_kd_tree(points, n, tree);

        int i,
            test_runs = n;

        long start_time = startTiming();
        for (i = 0; i < test_runs; ++i)
        {
            kNN(points[i], tree, n, k, result, stack_ptr, k_stack_ptr);
        }
        printf("Time = %ld ms, Size = %d Elements\n", endTiming(start_time), n);

        free(tree);
        free(result);
        free(points);
        free(stack_ptr);
        free(k_stack_ptr);
    };
};

TEST(kd_search_openmp, query_all_timing)
{
    int n, k = 5;

    for (n = 10000; n <= 10000; n += 1000)
    {
        struct Point *points = (struct Point *) malloc(n  * sizeof(struct Point));
        struct Node *tree = (struct Node *) malloc(n  * sizeof(struct Node));

        readPoints("../tests/data/10000_points.data", n, points);

        hipDeviceReset();
        build_kd_tree(points, n, tree);

        int test_runs = n;
        int *result = (int *) malloc(test_runs * k * sizeof(int));

        hipEvent_t start, stop;
        float elapsed_time = 0;
        int bytes = n * (sizeof(struct Node));

        cudaStartTiming(start, stop, elapsed_time);
        queryAll(points, tree, test_runs, n, k, result);
        cudaStopTiming(start, stop, elapsed_time);
        printCudaTiming(elapsed_time, bytes, n);

        free(tree);
        free(points);
        hipDeviceReset();
    };
};
