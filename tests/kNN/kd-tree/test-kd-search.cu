#include <float.h>
#include <math.h>

#include <test-common.cuh>

#include <kd-search.cuh>
#include <knn_gpgpu.h>

bool isExpectedPoint(struct Node *tree, int n, int k,  float qx, float qy, float qz, float ex, float ey, float ez)
{
    struct Point query_point;

    struct SPoint *s_stack_ptr = (struct SPoint *)malloc(51 * sizeof(struct SPoint));
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc((k + 1) * sizeof(KPoint));

    int *result = (int *) malloc(k * sizeof(int));

    query_point.p[0] = qx, query_point.p[1] = qy, query_point.p[2] = qz;

    cuKNN(query_point, tree, n, k, result, s_stack_ptr, k_stack_ptr);

    float actual = tree[result[0]].p[0] + tree[result[0]].p[1] + tree[result[0]].p[2];
    float expected = ex + ey + ez;

    // printf(">> WP tree\nsearching for (%3.1f, %3.1f, %3.1f)\n"
    //        "found (%3.1f, %3.1f, %3.1f) seen %d nodes\n\n",
    //        qx, qy, qz,
    //        tree[result[0]].p[0], tree[result[0]].p[1], tree[result[0]].p[2], visited);

    free(s_stack_ptr);
    free(k_stack_ptr);
    free(result);

    if (actual == expected)
    {
        return true;
    }
    return false;
}

TEST(kd_search, isEmpty)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(4 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1;

    cuInitStack(&stack);

    value1.index = 10;

    ASSERT_TRUE(cuIsEmpty(stack));

    stack[0] = value1;
    stack++;
    ASSERT_FALSE(cuIsEmpty(stack));

    free(stack_ptr);
}

TEST(kd_search, push)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(3 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1,
                    value2;

    cuInitStack(&stack);

    value1.index = 1;
    value2.index = 3;

    cuPush(&stack, value1);
    cuPush(&stack, value2);
    ASSERT_EQ(value1.index, stack_ptr[1].index);
    ASSERT_EQ(value2.index, stack_ptr[2].index);

    free(stack_ptr);
}

TEST(kd_search, pop)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(4 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1,
                    value2,
                    value3;

    cuInitStack(&stack);

    value1.index = 1;
    value2.index = 2;
    value3.index = 3;

    stack[0] = value1;
    stack[1] = value2;
    stack[2] = value3;
    stack += 3;

    ASSERT_EQ(value3.index, cuPop(&stack).index);
    ASSERT_EQ(value2.index, cuPop(&stack).index);
    ASSERT_EQ(value1.index, cuPop(&stack).index);

    free(stack_ptr);
}

TEST(kd_search, peek)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(4 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1;

    cuInitStack(&stack);

    value1.index = 10;

    ASSERT_EQ(-1, cuPeek(stack).index);
    ASSERT_EQ(-1, cuPeek(stack).index);

    cuPush(&stack, value1);

    ASSERT_EQ(value1.index, cuPeek(stack).index);
    ASSERT_EQ(value1.index, cuPeek(stack).index);

    free(stack_ptr);
}

TEST(kd_search, initKStack)
{
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc(51 * sizeof(KPoint)),
                   *k_stack = k_stack_ptr;

    cuInitKStack(&k_stack, 50);

    ASSERT_EQ(-1, k_stack[-1].dist);
    ASSERT_EQ(FLT_MAX, k_stack[0].dist);
    ASSERT_EQ(FLT_MAX, k_stack[49].dist);

    free(k_stack_ptr);
}

TEST(kd_search, insert)
{
    int n = 3;
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc(51 * sizeof(KPoint)),
                   *k_stack = k_stack_ptr;

    cuInitKStack(&k_stack, n);
    struct KPoint a, b, c, d;

    a.dist = 1;
    b.dist = 2;
    c.dist = 3;
    d.dist = 0;

    cuInsert(k_stack, a, n);
    ASSERT_EQ(FLT_MAX, cuLook(k_stack, n).dist);
    ASSERT_EQ(a.dist, k_stack[0].dist);

    cuInsert(k_stack, b, n);
    ASSERT_EQ(FLT_MAX, cuLook(k_stack, n).dist);
    ASSERT_EQ(b.dist, k_stack[1].dist);

    cuInsert(k_stack, c, n);
    ASSERT_EQ(c.dist, cuLook(k_stack, n).dist);
    ASSERT_EQ(c.dist, k_stack[2].dist);

    cuInsert(k_stack, d, n);
    ASSERT_EQ(b.dist, cuLook(k_stack, n).dist);
    ASSERT_EQ(d.dist, k_stack[0].dist);

    free(k_stack_ptr);
}

TEST(kd_search, insert_k_is_one)
{
    int n = 1;
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc(51 * sizeof(KPoint)),
                   *k_stack = k_stack_ptr;

    cuInitKStack(&k_stack, n);
    struct KPoint a, b;

    a.dist = 1;
    b.dist = 0;

    cuInsert(k_stack, a, n);
    ASSERT_EQ(a.dist, cuLook(k_stack, n).dist);
    ASSERT_EQ(a.dist, k_stack[0].dist);

    cuInsert(k_stack, b, n);
    ASSERT_EQ(b.dist, cuLook(k_stack, n).dist);
    ASSERT_EQ(b.dist, k_stack[0].dist);

    free(k_stack_ptr);
}

TEST(kd_search, upDim)
{
    int dim = 0;

    cuUpDim(&dim);
    ASSERT_EQ(1, dim);

    cuUpDim(&dim);
    ASSERT_EQ(2, dim);

    cuUpDim(&dim);
    ASSERT_EQ(0, dim);

    cuUpDim(&dim);
    ASSERT_EQ(1, dim);
}

TEST(kd_search, correctness_with_k)
{
    int n = 6,
        k = 3;

    int *result = (int *) malloc(k * sizeof(int));

    struct Point *points = (struct Point *) malloc(n  * sizeof(Point));
    struct Node *tree = (struct Node *) malloc(n  * sizeof(Node));

    struct SPoint *s_stack_ptr = (struct SPoint *)malloc(51 * sizeof(struct SPoint));
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc((k + 1) * sizeof(KPoint));

    points[0].p[0] = 2, points[0].p[1] = 3, points[0].p[2] = 0;
    points[1].p[0] = 5, points[1].p[1] = 4, points[1].p[2] = 0;
    points[2].p[0] = 9, points[2].p[1] = 6, points[2].p[2] = 0;
    points[3].p[0] = 4, points[3].p[1] = 7, points[3].p[2] = 0;
    points[4].p[0] = 8, points[4].p[1] = 1, points[4].p[2] = 0;
    points[5].p[0] = 7, points[5].p[1] = 2, points[5].p[2] = 0;

    hipDeviceReset();
    buildKdTree(points, n, tree);

    hipDeviceReset();
    cuKNN(points[4], tree, n, k, result, s_stack_ptr, k_stack_ptr);

    ASSERT_EQ(4, result[0]);
    ASSERT_EQ(3, result[1]);
    ASSERT_EQ(1, result[2]);

    free(points);
    free(tree);

    free(s_stack_ptr);
    free(k_stack_ptr);
}

TEST(kd_search, correctness_with_10000_points_file)
{
    int n, k = 1;

    for (n = 1000; n <= 10000; n += 1000)
    {
        struct Point *points = (struct Point *) malloc(n  * sizeof(Point));
        struct Node *tree = (struct Node *) malloc(n  * sizeof(Node));

        srand((int)time(NULL));

        readPoints("../tests/data/10000_points.data", n, points);

        hipDeviceReset();
        buildKdTree(points, n, tree);

        // printTree(tree, 0, n / 2);

        int *result = (int *) malloc(k * sizeof(int));

        int i,
            test_runs = n;

        struct SPoint *stack_ptr = (struct SPoint *)malloc(51 * sizeof(struct SPoint));
        struct KPoint *k_stack_ptr = (struct KPoint *) malloc((k + 1) * sizeof(KPoint));

        for (i = 0; i < test_runs; ++i)
        {
            hipDeviceReset();
            cuKNN(points[i], tree, n, k, result, stack_ptr, k_stack_ptr);

            // printf("Looking for (%3.1f, %3.1f, %3.1f), found (%3.1f, %3.1f, %3.1f)\n",
            //        tree[i].p[0], tree[i].p[1], tree[i].p[2],
            //        tree[result[0]].p[0], tree[result[0]].p[1], tree[result[0]].p[2]);

            ASSERT_EQ(points[i].p[0], tree[result[0]].p[0]) << "Failed at i = " << i << " with n = " << n ;
            ASSERT_EQ(points[i].p[1], tree[result[0]].p[1]) << "Failed at i = " << i << " with n = " << n;
            ASSERT_EQ(points[i].p[2], tree[result[0]].p[2]) << "Failed at i = " << i << " with n = " << n;
        }

        free(tree);
        free(result);
        free(points);
        free(stack_ptr);
        free(k_stack_ptr);
    };
};

TEST(kd_search, cu_query_all_correctness_with_10000_points_file)
{
    int n, i, k = 50;

    for (n = 1000; n <= 100000; n += 100000)
    {
        struct Point *points = (struct Point *) malloc(n  * sizeof(Point));
        struct Node *tree = (struct Node *) malloc(n  * sizeof(Node));

        srand((int)time(NULL));

        if (n > 10000)
        {
            populatePointSRosetta(points,  n);
            // readPoints("/home/simenhg/workspace/tsi-gpgpu/tests/data/100_mill_points.data", n, points);
        }
        else
        {
            readPoints("../tests/data/10000_points.data", n, points);
        }

        hipDeviceReset();
        buildKdTree(points, n, tree);
        // printTree(tree, 0, n / 2);

        int *result = (int *) malloc(n * k * sizeof(int));

        cuQueryAll(points, tree, n, n, k, result);
        for (i = 0; i < n; ++i)
        {
            ASSERT_GT(result[i * k], -1) << "Result index is less then 0 \n Failed at i = " << i << " with n = " << n ;
            ASSERT_LT(result[i * k], n) << "Result index is bigger then the length of the tree \n Failed at i = " << i << " with n = " << n ;
            ASSERT_EQ(points[i].p[0], tree[result[i * k]].p[0]) << "Failed at i = " << i << " with n = " << n ;
            ASSERT_EQ(points[i].p[1], tree[result[i * k]].p[1]) << "Failed at i = " << i << " with n = " << n;
            ASSERT_EQ(points[i].p[2], tree[result[i * k]].p[2]) << "Failed at i = " << i << " with n = " << n;
        }

        free(tree);
        free(result);
        free(points);
    };
};

TEST(kd_search, knn_wikipedia_example)
{
    int n = 6,
        k = 1;

    struct Point *points = (struct Point *) malloc(n  * sizeof(Point));
    struct Node *tree = (struct Node *) malloc(n  * sizeof(Node));

    points[0].p[0] = 2, points[0].p[1] = 3, points[0].p[2] = 0;
    points[1].p[0] = 5, points[1].p[1] = 4, points[1].p[2] = 0;
    points[2].p[0] = 9, points[2].p[1] = 6, points[2].p[2] = 0;
    points[3].p[0] = 4, points[3].p[1] = 7, points[3].p[2] = 0;
    points[4].p[0] = 8, points[4].p[1] = 1, points[4].p[2] = 0;
    points[5].p[0] = 7, points[5].p[1] = 2, points[5].p[2] = 0;

    hipDeviceReset();
    buildKdTree(points, n, tree);


    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 2, 3, 0, 2, 3, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 5, 4, 0, 5, 4, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 9, 6, 0, 9, 6, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 4, 7, 0, 4, 7, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 8, 1, 0, 8, 1, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 7, 2, 0, 7, 2, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 10, 10, 0, 9, 6, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 0, 0, 0, 2, 3, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 4, 4, 0, 5, 4, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 3, 2, 0, 2, 3, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 2, 6, 0, 4, 7, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 10, 0, 0, 8, 1, 0));
    ASSERT_EQ(true, isExpectedPoint(tree, n, k, 0, 10, 0, 4, 7, 0));

    free(points);
    free(tree);
}

TEST(kd_search, cu_query_all_wikipedia_example)
{
    int n = 6, k = 1;
    struct Point *points = (struct Point *) malloc(n * sizeof(Point));
    struct Node *tree = (struct Node *) malloc(n * sizeof(Node));
    int *result = (int *) malloc(n * k * sizeof(int));

    points[0].p[0] = 2, points[0].p[1] = 3, points[0].p[2] = 0;
    points[1].p[0] = 5, points[1].p[1] = 4, points[1].p[2] = 0;
    points[2].p[0] = 9, points[2].p[1] = 6, points[2].p[2] = 0;
    points[3].p[0] = 4, points[3].p[1] = 7, points[3].p[2] = 0;
    points[4].p[0] = 8, points[4].p[1] = 1, points[4].p[2] = 0;
    points[5].p[0] = 7, points[5].p[1] = 2, points[5].p[2] = 0;

    hipDeviceReset();
    buildKdTree(points, n, tree);
    cuQueryAll(points, tree, n, n, 1, result);

    ASSERT_EQ(result[0], 0);
    ASSERT_EQ(result[1], 1);
    ASSERT_EQ(result[2], 5);
    ASSERT_EQ(result[3], 2);
    ASSERT_EQ(result[4], 4);
    ASSERT_EQ(result[5], 3);

    free(points);
    free(tree);
    free(result);
}

TEST(kd_search, knn_timing)
{
    int n, k = 1;

    for (n = 10000; n <= 10000; n += 10000)
    {
        struct Point *points = (struct Point *) malloc(n  * sizeof(Point));
        struct Node *tree = (struct Node *) malloc(n  * sizeof(Node));

        struct SPoint *stack_ptr = (struct SPoint *)malloc(51 * sizeof(struct SPoint));
        struct KPoint *k_stack_ptr = (struct KPoint *) malloc((k + 1) * sizeof(KPoint));

        int *result = (int *) malloc(k * sizeof(int));

        srand((int)time(NULL));

        readPoints("../tests/data/10000_points.data", n, points);

        hipDeviceReset();
        buildKdTree(points, n, tree);

        int i,
            test_runs = n;

        hipDeviceReset();

        hipEvent_t start, stop;
        float elapsed_time = 0;
        int bytes = n * (sizeof(Node));

        cudaStartTiming(start, stop, elapsed_time);

        long start_time = startTiming();
        for (i = 0; i < test_runs; ++i)
        {
            cuKNN(points[i], tree, n, k, result, stack_ptr, k_stack_ptr);
        }

        cudaStopTiming(start, stop, elapsed_time);
        printCudaTiming(elapsed_time, bytes, n);

        free(tree);
        free(result);
        free(points);
        free(stack_ptr);
        free(k_stack_ptr);
    };
};

TEST(kd_search, query_all_timing)
{
    int n, k = 5;

    for (n = 10000; n <= 10000; n += 10000)
    {
        struct Point *points = (struct Point *) malloc(n  * sizeof(Point));
        struct Node *tree = (struct Node *) malloc(n  * sizeof(Node));

        if (n > 10000)
        {
            populatePointSRosetta(points,  n);
            // readPoints("/home/simenhg/workspace/tsi-gpgpu/tests/data/100_mill_points.data", n, points);
        }
        else
        {
            readPoints("../tests/data/10000_points.data", n, points);

        }

        hipDeviceReset();
        buildKdTree(points, n, tree);

        int test_runs = n;
        int *result = (int *) malloc(test_runs * k * sizeof(int));

        hipEvent_t start, stop;
        float elapsed_time = 0;
        int bytes = n * (sizeof(Node));

        cudaStartTiming(start, stop, elapsed_time);
        cuQueryAll(points, tree, test_runs, n, k, result);
        cudaStopTiming(start, stop, elapsed_time);
        printCudaTiming(elapsed_time, bytes, n);

        free(tree);
        free(points);
        hipDeviceReset();
    };
};
