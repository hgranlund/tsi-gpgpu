#include "hip/hip_runtime.h"
#include <search-iterative.cuh>
#include <knn_gpgpu.h>
#include <float.h>
#include "math.h"
#include <sys/time.h>

#include "test-common.cuh"

bool isExpectedPoint(struct Point *tree, int n, int k,  float qx, float qy, float qz, float ex, float ey, float ez)
{
    struct Point query_point;
    int result[k], visited = 0;
    query_point.p[0] = qx, query_point.p[1] = qy, query_point.p[2] = qz;

    kNN(query_point, tree, n, k, result, &visited);
    float actual = tree[result[0]].p[0] + tree[result[0]].p[1] + tree[result[0]].p[2];
    float expected = ex + ey + ez;

    // printf(">> WP tree\nsearching for (%3.1f, %3.1f, %3.1f)\n"
    //        "found (%3.1f, %3.1f, %3.1f) seen %d nodes\n\n",
    //        qx, qy, qz,
    //        tree[result[0]].p[0], tree[result[0]].p[1], tree[result[0]].p[2], visited);

    if (actual == expected)
    {
        return true;
    }
    return false;
}

void printTree(struct Point *tree, int level, int root)
{
    if (root < 0) return;

    int i;

    printf("|");
    for (i = 0; i < level; ++i)
    {
        printf("----");
    }
    printf("(%3.1f, %3.1f, %3.1f)\n", tree[root].p[0], tree[root].p[1], tree[root].p[2]);

    printTree(tree, 1 + level, tree[root].left);
    printTree(tree, 1 + level, tree[root].right);
}

TEST(search_iterative, push)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(3 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1,
                    value2;

    initStack(&stack);

    value1.index = 1;
    value2.index = 3;

    push(&stack, value1);
    push(&stack, value2);
    ASSERT_EQ(value1.index, stack_ptr[1].index);
    ASSERT_EQ(value2.index, stack_ptr[2].index);

    free(stack_ptr);
}

TEST(search_iterative, pop)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(4 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1,
                    value2,
                    value3;

    initStack(&stack);

    value1.index = 1;
    value2.index = 2;
    value3.index = 3;

    stack[0] = value1;
    stack[1] = value2;
    stack[2] = value3;
    stack += 3;

    ASSERT_EQ(value3.index, pop(&stack).index);
    ASSERT_EQ(value2.index, pop(&stack).index);
    ASSERT_EQ(value1.index, pop(&stack).index);

    free(stack_ptr);
}

TEST(search_iterative, isEmpty)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(4 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1;

    initStack(&stack);

    value1.index = 10;

    ASSERT_TRUE(isEmpty(stack));

    stack[0] = value1;
    stack++;
    ASSERT_FALSE(isEmpty(stack));

    free(stack_ptr);
}

TEST(search_iterative, peek)
{
    struct SPoint *stack_ptr = (struct SPoint *) malloc(4 * sizeof(struct SPoint)),
                   *stack = stack_ptr,
                    value1;

    initStack(&stack);

    value1.index = 10;

    ASSERT_EQ(-1, peek(stack).index);
    ASSERT_EQ(-1, peek(stack).index);

    push(&stack, value1);

    ASSERT_EQ(value1.index, peek(stack).index);
    ASSERT_EQ(value1.index, peek(stack).index);

    free(stack_ptr);
}

TEST(search_iterative, upDim)
{
    int dim = 0;

    upDim(&dim);
    ASSERT_EQ(1, dim);

    upDim(&dim);
    ASSERT_EQ(2, dim);

    upDim(&dim);
    ASSERT_EQ(0, dim);

    upDim(&dim);
    ASSERT_EQ(1, dim);
}


TEST(search_iterative, initKStack)
{
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc(51 * sizeof(KPoint)),
                   *k_stack = k_stack_ptr;

    initKStack(&k_stack, 50);

    ASSERT_EQ(-1, k_stack[-1].dist);
    ASSERT_EQ(FLT_MAX, k_stack[0].dist);
    ASSERT_EQ(FLT_MAX, k_stack[49].dist);

    free(k_stack_ptr);
}

TEST(search_iterative, insert)
{
    int n = 3;
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc(51 * sizeof(KPoint)),
                   *k_stack = k_stack_ptr;

    initKStack(&k_stack, n);
    struct KPoint a, b, c, d;

    a.dist = 1;
    b.dist = 2;
    c.dist = 3;
    d.dist = 0;

    insert(k_stack, a, n);
    ASSERT_EQ(FLT_MAX, look(k_stack, n).dist);
    ASSERT_EQ(a.dist, k_stack[0].dist);

    insert(k_stack, b, n);
    ASSERT_EQ(FLT_MAX, look(k_stack, n).dist);
    ASSERT_EQ(b.dist, k_stack[1].dist);

    insert(k_stack, c, n);
    ASSERT_EQ(c.dist, look(k_stack, n).dist);
    ASSERT_EQ(c.dist, k_stack[2].dist);

    insert(k_stack, d, n);
    ASSERT_EQ(b.dist, look(k_stack, n).dist);
    ASSERT_EQ(d.dist, k_stack[0].dist);

    free(k_stack_ptr);
}

TEST(search_iterative, insert_k_is_one)
{
    int n = 1;
    struct KPoint *k_stack_ptr = (struct KPoint *) malloc(51 * sizeof(KPoint)),
                   *k_stack = k_stack_ptr;

    initKStack(&k_stack, n);
    struct KPoint a, b;

    a.dist = 1;
    b.dist = 0;

    insert(k_stack, a, n);
    ASSERT_EQ(a.dist, look(k_stack, n).dist);
    ASSERT_EQ(a.dist, k_stack[0].dist);

    insert(k_stack, b, n);
    ASSERT_EQ(b.dist, look(k_stack, n).dist);
    ASSERT_EQ(b.dist, k_stack[0].dist);

    free(k_stack_ptr);
}

TEST(search_iterative, wikipedia_example)
{
    int n = 6,
        k = 1;

    struct PointS *points = (struct PointS *) malloc(n  * sizeof(PointS));
    struct Point *points_out = (struct Point *) malloc(n  * sizeof(Point));

    points[0].p[0] = 2, points[0].p[1] = 3, points[0].p[2] = 0;
    points[1].p[0] = 5, points[1].p[1] = 4, points[1].p[2] = 0;
    points[2].p[0] = 9, points[2].p[1] = 6, points[2].p[2] = 0;
    points[3].p[0] = 4, points[3].p[1] = 7, points[3].p[2] = 0;
    points[4].p[0] = 8, points[4].p[1] = 1, points[4].p[2] = 0;
    points[5].p[0] = 7, points[5].p[1] = 2, points[5].p[2] = 0;

    hipDeviceReset();
    build_kd_tree(points, n, points_out);


    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 2, 3, 0, 2, 3, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 5, 4, 0, 5, 4, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 9, 6, 0, 9, 6, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 4, 7, 0, 4, 7, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 8, 1, 0, 8, 1, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 7, 2, 0, 7, 2, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 10, 10, 0, 9, 6, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 0, 0, 0, 2, 3, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 4, 4, 0, 5, 4, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 3, 2, 0, 2, 3, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 2, 6, 0, 4, 7, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 10, 0, 0, 8, 1, 0));
    ASSERT_EQ(true, isExpectedPoint(points_out, n, k, 0, 10, 0, 4, 7, 0));

    free(points);
    free(points_out);
}

TEST(search_iterative, correctness_with_k)
{
    int n = 6,
        k = 3,
        result[k],
        visited;

    struct PointS *points = (struct PointS *) malloc(n  * sizeof(PointS));
    struct Point *points_out = (struct Point *) malloc(n  * sizeof(Point));

    points[0].p[0] = 2, points[0].p[1] = 3, points[0].p[2] = 0;
    points[1].p[0] = 5, points[1].p[1] = 4, points[1].p[2] = 0;
    points[2].p[0] = 9, points[2].p[1] = 6, points[2].p[2] = 0;
    points[3].p[0] = 4, points[3].p[1] = 7, points[3].p[2] = 0;
    points[4].p[0] = 8, points[4].p[1] = 1, points[4].p[2] = 0;
    points[5].p[0] = 7, points[5].p[1] = 2, points[5].p[2] = 0;

    hipDeviceReset();
    build_kd_tree(points, n, points_out);
    kNN(points_out[4], points_out, n, k, result, &visited);

    ASSERT_EQ(4, result[0]);
    ASSERT_EQ(3, result[1]);
    ASSERT_EQ(1, result[2]);

    free(points);
    free(points_out);
}

double WallTime ()
{
    struct timeval tmpTime;
    gettimeofday(&tmpTime, NULL);
    return tmpTime.tv_sec + tmpTime.tv_usec / 1.0e6;
}

int midpoint(int lower, int upper)
{
    return (int) floor((upper - lower) / 2) + lower;
}

struct kd_node_t
{
    float x[3];
    struct kd_node_t *left, *right;
};

int convertTree(struct kd_node_t *root, struct Point *tree, int lower, int upper)
{
    if (!root) return -1;

    int index = midpoint(lower, upper);

    struct Point temp;

    temp.p[0] = root->x[0];
    temp.p[1] = root->x[1];
    temp.p[2] = root->x[2];

    temp.left = convertTree(root->left, tree, lower, index);
    temp.right = convertTree(root->right, tree, index + 1, upper);

    tree[index] = temp;

    return index;
}

inline float dist(struct kd_node_t *a, struct kd_node_t *b, int dim)
{
    float t,
          d = 0;
    while (dim--)
    {
        t = a->x[dim] - b->x[dim];
        d += t * t;
    }
    return d;
}

void swapIt(struct kd_node_t *x, struct kd_node_t *y)
{
    float tmp[3];
    memcpy(tmp,  x->x, sizeof(tmp));
    memcpy(x->x, y->x, sizeof(tmp));
    memcpy(y->x, tmp,  sizeof(tmp));
}

/* see quickselect method */
struct kd_node_t *findMedian(struct kd_node_t *start, struct kd_node_t *end, int idx)
{
    if (end <= start) return NULL;
    if (end == start + 1)
        return start;

    struct kd_node_t *p, *store, *md = start + (end - start) / 2;
    float pivot;
    while (1)
    {
        pivot = md->x[idx];

        swapIt(md, end - 1);
        for (store = p = start; p < end; p++)
        {
            if (p->x[idx] < pivot)
            {
                if (p != store)
                    swapIt(p, store);
                store++;
            }
        }
        swapIt(store, end - 1);

        /* median has duplicate values */
        if (store->x[idx] == md->x[idx])
            return md;

        if (store > md) end = store;
        else        start = store;
    }
}

struct kd_node_t *makeTree(struct kd_node_t *t, int len, int i, int dim)
{
    struct kd_node_t *n;

    if (!len) return 0;

    if ((n = findMedian(t, t + len, i)))
    {
        i = (i + 1) % dim;
        n->left  = makeTree(t, n - t, i, dim);
        n->right = makeTree(n + 1, t + len - (n + 1), i, dim);
    }
    return n;
}

void read_points(const char *file_path, int n, kd_node_t *points)
{
    FILE *file = fopen(file_path, "rb");
    if (file == NULL)
    {
        fputs ("File error\n", stderr);
        exit (1);
    }
    for (int i = 0; i < n; ++i)
    {
        fread(&points[i].x, sizeof(float), 3, file);
        for (int j = 0; j < 3; ++j)
        {
            points[i].x[j] = round(points[i].x[j] / 100000000.0);
        }
    }

    fclose(file);
}

TEST(search_iterative, timing)
{
    int n, k = 1;

    for (n = 1000; n <= 10000; n += 1000)
    {
        struct PointS *points = (struct PointS *) malloc(n  * sizeof(PointS));
        struct Point *points_out = (struct Point *) malloc(n  * sizeof(Point));
        struct Point *qp_points = (struct Point *) malloc(n  * sizeof(Point));
        srand(time(NULL));

        readPoints("../tests/data/10000_points.data", n, points);

        for (int i = 0; i < n; ++i)
        {
            struct Point point;
            point.p[0] = points[i].p[0];
            point.p[1] = points[i].p[1];
            point.p[2] = points[i].p[2];
            qp_points[i] = point;
            // points_out[i] = point;
        }

        build_kd_tree(points, n, points_out);

        // struct kd_node_t *million = (struct kd_node_t *) calloc(n, sizeof(struct kd_node_t));

        // read_points("/home/simenhg/workspace/tsi-gpgpu/tests/data/10000_points.data", n, million);

        // struct kd_node_t *root = makeTree(million, n, 0, 3);

        // convertTree(root, points_out, 0, n);

        // printTree(points_out, 0, n / 2);

        int *result = (int *) malloc(k * sizeof(int));

        int i,
            visited = 0,
            sum = 0,
            test_runs = n;

        struct SPoint *stack_ptr = (struct SPoint *)malloc(51 * sizeof(struct SPoint));
        struct KPoint *k_stack_ptr = (struct KPoint *) malloc((k + 1) * sizeof(KPoint));

        double start_time = WallTime();
        for (i = 0; i < test_runs; ++i)
        {
            visited = 0;
            kNN(points_out[i], points_out, n, k, result, &visited, stack_ptr, k_stack_ptr);
            sum += visited;

            // printf("Looking for (%3.1f, %3.1f, %3.1f), found (%3.1f, %3.1f, %3.1f)\n",
            //        points_out[i].p[0], points_out[i].p[1], points_out[i].p[2],
            //        points_out[result[0]].p[0], points_out[result[0]].p[1], points_out[result[0]].p[2]);

            ASSERT_EQ(points_out[i].p[0], points_out[result[0]].p[0]) << "Failed at i = " << i << " with n = " << n ;
            ASSERT_EQ(points_out[i].p[1], points_out[result[0]].p[1]) << "Failed at i = " << i << " with n = " << n;
            ASSERT_EQ(points_out[i].p[2], points_out[result[0]].p[2]) << "Failed at i = " << i << " with n = " << n;
        }

        // printf("Time = %lf ms, Size = %d Elements, Awg visited = %3.1f\n", ((WallTime() - start_time) * 1000), n, sum / (float)test_runs);

        free(points_out);
        free(result);
        free(points);
    };
};


// #include "test-common.cuh"
// #include <knn_gpgpu.h>

// TEST(kd_search, wikipedia_example)
// {
//     int n = 6, k = 1;
//     struct PointS *points = (struct PointS *) malloc(n * sizeof(PointS));
//     struct Point *points_out = (struct Point *) malloc(n * sizeof(Point));
//     int *result = (int *) malloc(n * k * sizeof(int));

//     points[0].p[0] = 2, points[0].p[1] = 3, points[0].p[2] = 0;
//     points[1].p[0] = 5, points[1].p[1] = 4, points[1].p[2] = 0;
//     points[2].p[0] = 9, points[2].p[1] = 6, points[2].p[2] = 0;
//     points[3].p[0] = 4, points[3].p[1] = 7, points[3].p[2] = 0;
//     points[4].p[0] = 8, points[4].p[1] = 1, points[4].p[2] = 0;
//     points[5].p[0] = 7, points[5].p[1] = 2, points[5].p[2] = 0;

//     hipDeviceReset();
//     build_kd_tree(points, n, points_out);
//     queryAll(points_out, points_out, n, n, 1, result);

//     for (int i = 0; i < n; ++i)
//     {
//         ASSERT_EQ(result[i], i);
//     }

//     free(points);
//     free(points_out);
//     free(result);
// }

// TEST(kd_search, timing)
// {
//     int n, k = 1;

//     for (n = 32; n <= 32; n += 250000)
//     {
//         struct PointS *points = (struct PointS *) malloc(n  * sizeof(PointS));
//         struct Point *points_out = (struct Point *) malloc(n  * sizeof(Point));
//         srand(time(NULL));

//         populatePointSs(points, n);

//         build_kd_tree(points, n, points_out);

//         int test_runs = n;
//         int *result = (int *) malloc(test_runs * k * sizeof(int));
//         struct Point *query_data = (struct Point *) malloc(test_runs * sizeof(Point));

//         populatePoints(query_data, n);

//         hipEvent_t start, stop;
//         float elapsed_time = 0;
//         int bytes = n * (sizeof(Point));

//         cudaStartTiming(start, stop, elapsed_time);
//         queryAll(query_data, points_out, test_runs, n, k, result);
//         cudaStopTiming(start, stop, elapsed_time);
//         printCudaTiming(elapsed_time, bytes, n);

//         free(query_data);
//         free(points_out);
//         free(points);
//         hipDeviceReset();
//     };
// };
