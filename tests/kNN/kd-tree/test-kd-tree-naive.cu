#include <kd-tree-naive.cuh>
#include <knn_gpgpu.h>

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include "gtest/gtest.h"


#define debug 0

void print_tree(float *tree, int level, int lower, int upper, int n)
{
  if (debug)
  {
    if (lower >= upper)
    {
      return;
    }

    int i, r = midpoint(lower, upper);

    printf("|");
    for (i = 0; i < level; ++i)
    {
      printf("--");
    }
    printf("(%3.1f, %3.1f, %3.1f)\n", tree[h_index(r, 0, n)], tree[h_index(r, 1, n)], tree[h_index(r, 2, n)]);

    print_tree(tree, 1 + level, lower, r, n);
    print_tree(tree, 1 + level, r + 1, upper, n);
  }
}

TEST(kd_tree_naive, kd_tree_naive_correctness){
  int i, j, n = 1000;
  float *points, *expected_points;
  points = (float*) malloc(n * 3 * sizeof(float));
  expected_points = (float*) malloc(n * 3 * sizeof(float));
  srand(time(NULL));
  for ( i = 0; i < n; ++i)
  {
    for ( j = 0; j < 3; ++j)
    {
      points[h_index(i, j, n)] = n - i -1 +j;
      points[h_index(i, j, n)] = (float) rand() /100000000;
      expected_points[h_index(i, j, n)] = i ;
    }
  }
  if (debug)
  {


    printf("kd tree:\n");
    print_tree(points, 0, 0, n, n);
    printf("==================\n");

  }

  build_kd_tree(points, n);

  for ( i = 0; i < n; ++i)
  {
    for ( j = 0; j < 3; ++j)
    {
// ASSERT_EQ(points[h_index(i, j, 5n)] ,i) << "Faild with i = " << i << " j = " <<j ;
    }
  }

  if (debug)
  {

    printf("kd tree:\n");
    print_tree(points, 0, 0, n, n);
    printf("==================\n");

  }

  free(points);
  free(expected_points);
}

TEST(kd_tree_naive, kd_tree_naive_timeing)
{
  int i, j, n = 65536;
  float *points;
  points = (float*) malloc(n * 3 * sizeof(float));
  srand(time(NULL));
  for ( i = 0; i < n; ++i)
  {
    for ( j = 0; j < 3; ++j)
    {
      points[h_index(i, j, n)] = (float) rand() /100000000;
    }
  }

  hipEvent_t start, stop;
  unsigned int bytes = n*3 * (sizeof(float));
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  float elapsed_time=0;

  checkCudaErrors(hipEventRecord(start, 0));

  build_kd_tree(points, n);

  checkCudaErrors(hipEventRecord(stop, 0));
  hipEventSynchronize(start);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);
  elapsed_time = elapsed_time ;
  double throughput = 1.0e-9 * ((double)bytes)/(elapsed_time* 1e-3);
  printf("build_kd_tree_naive, Throughput = %.4f GB/s, Time = %.5f ms, Size = %u Elements, NumDevsUsed = %d\n",
    throughput, elapsed_time, n, 1);

  free(points);
}


// float *ref, *dist;
// float *query;
// int *ind;
// unsigned int    ref_nb = 131072;
// unsigned int    query_nb = 1;
// unsigned int    dim=3;
// unsigned int    k          = 100;
// unsigned int    iterations = 1;
// unsigned int    i;

// ref    = (float *) malloc(ref_nb   * dim * sizeof(float));
// query  = (float *) malloc(query_nb * dim * sizeof(float));
// dist  = (float *) malloc(k * sizeof(float));
// ind  = (int *) malloc(k * sizeof(float));

// for (unsigned int count = 0; count < ref_nb*dim; count++)
// {
//   ref[count] = (float)ref_nb*dim-count;
// }
// for (unsigned int count = 0; count < query_nb*dim; count++)
// {
//   query[count] = 0;
// }

// for (i=0; i<iterations; i++){
//   knn_brute_force_reduce(ref, ref_nb, query, dim, k, dist, ind);
// }

// for (unsigned int i = 0; i < k; ++i)
// {
//   ASSERT_EQ(ind[i], ref_nb-1-i) << "Faild with i = "<<i << " and n = " << ref_nb;;
// }

// free(dist);
// free(ind);
// free(query);
// free(ref);
// hipDeviceSynchronize();
// hipDeviceReset();
