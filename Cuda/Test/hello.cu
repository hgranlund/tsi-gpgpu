
#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 10

__global__ void add(int *a, int *b, int *c)
{
  int tID = blockIdx.x;
  if (tID<N)
    {
      c[tID] = a[tID] + b[tID];
    }
}
int main()
{
  int a[N], b[N], c[N];
  int *d_a, *d_b, *d_c;


  hipMalloc((void **) &d_a, N*sizeof(int));
  hipMalloc((void **) &d_b, N*sizeof(int));
  hipMalloc((void **) &d_c, N*sizeof(int));

  for (int i = 0; i < N; i++)
  {
    a[i] = i,
    b[i] = 1;
  }
hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);
hipMemcpy(d_c, c, N*sizeof(int), hipMemcpyHostToDevice);

  add<<<N,1>>> (d_a, d_b, d_c);

hipMemcpy(c,d_c, N*sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < N; i++)
  {
    printf("%d + %d = %d\n", a[i], b[i], c[i]);
  }
  return 0;
}
