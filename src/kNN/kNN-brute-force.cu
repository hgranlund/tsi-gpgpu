#include "hip/hip_runtime.h"


// Includes
#include <kNN-brute-force.cuh>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "bitonic-sort-serial.c"

__constant__  float query_dev[3];

__global__ void cuComputeDistanceGlobal( float* ref, int ref_nb , int dim,  float* dist, int* ind){

  // restiction: dim=3
  float dx,dy,dz;

  int index = blockIdx.x * dim;
  while (index < ref_nb){
    dx=ref[index] - query_dev[0];
    dy=ref[index + 1] - query_dev[1];
    dz=ref[index + 2] - query_dev[2];
    dist[index/dim] = (dx*dx)+(dy*dy)+(dz*dz);
    ind[index/dim] = index/dim;
    index += gridDim.x * dim;
  }
}


__global__ void cuParallelSqrt(float *dist, int k){
  unsigned int xIndex = blockIdx.x;
  if (xIndex < k){
    dist[xIndex] = sqrt(dist[xIndex]);
  }
}



void knn_brute_force(float* ref_host, int ref_nb, float* query_host, int dim, int k, float* dist_host, int* ind_host){

  unsigned int size_of_float = sizeof(float);
  unsigned int size_of_int   = sizeof(int);

  float        *ref_dev;
  float        *dist_dev;
  int          *ind_dev;


  hipMalloc( (void **) &dist_dev, ref_nb * size_of_float);
  hipMalloc( (void **) &ind_dev, ref_nb * size_of_int);
  hipMalloc( (void **) &ref_dev, ref_nb * size_of_float * dim);


  hipMemcpy(ref_dev, ref_host, ref_nb*dim*size_of_float, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(query_dev), query_host, dim*size_of_float);

  cuComputeDistanceGlobal<<<256,1>>>(ref_dev, ref_nb, dim, dist_dev, ind_dev);

  // cuParallelSqrt<<<k,1>>>(dist_dev, k);
  hipMemcpy(dist_host, dist_dev, k*size_of_float, hipMemcpyDeviceToHost);
  hipMemcpy(ind_host,  ind_dev,  k*size_of_int, hipMemcpyDeviceToHost);

  printf("\n[");
  for (int i = 0; i < k; ++i)
  {
    printf("%d, ",ind_host[i] );
  }
  printf("]\n");


  printf("\n[");
  for (int i = 0; i < k; ++i)
  {
    printf("%f, ",dist_host[i]);
  }
  printf("]\n");




  // bitonic_sort_serial(dist_host, k);

  hipFree(ref_dev);
  hipFree(ind_dev);
  hipFree(query_dev);
}
