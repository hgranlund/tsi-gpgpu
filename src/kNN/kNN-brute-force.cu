#include "hip/hip_runtime.h"


// Includes
#include <kNN-brute-force.cuh>
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include <time.h>


// Constants used by the program
#define MAX_PITCH_VALUE_IN_BYTES       262144
#define MAX_TEXTURE_WIDTH_IN_BYTES     65536
#define MAX_TEXTURE_HEIGHT_IN_BYTES    32768
#define MAX_PART_OF_FREE_MEMORY_USED   0.9
#define BLOCK_DIM                      16

#ifndef max
#define max(a,b) (((a) (b)) ? (a) : (b))
#define min(a,b) (((a) < (b)) ? (a) : (b))
#endif


// Texture containing the reference points (if it is possible)
texture<float, 2, hipReadModeElementType> texA;

__global__ void cuComputeDistanceTexture(int wA, float * B, int wB, int pB, int dim, float* AB){
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if ( xIndex<wB && yIndex<wA ){
    float ssd = 0;
    for (int i=0; i<dim; i++){
      float tmp  = tex2D(texA, (float)yIndex, (float)i) - B[ i * pB + xIndex ];
      ssd += tmp * tmp;
    }
    AB[yIndex * pB + xIndex] = ssd;
  }
}

__global__ void cuComputeDistanceGlobal( float* A, int wA, int pA, float* B, int wB, int pB, int dim,  float* AB){

  // Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
  __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
  __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

  // Sub-matrix of A (begin, step, end) and Sub-matrix of B (begin, step)
  __shared__ int begin_A;
  __shared__ int begin_B;
  __shared__ int step_A;
  __shared__ int step_B;
  __shared__ int end_A;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // Other variables
  float tmp;
  float ssd = 0;

  // Loop parameters
  begin_A = BLOCK_DIM * blockIdx.y;
  begin_B = BLOCK_DIM * blockIdx.x;
  step_A  = BLOCK_DIM * pA;
  step_B  = BLOCK_DIM * pB;
  end_A   = begin_A + (dim-1) * pA;

  // Conditions
  int cond0 = (begin_A + tx < wA); // used to write in shared memory
  int cond1 = (begin_B + tx < wB); // used to write in shared memory & to computations and to write in output matrix
  int cond2 = (begin_A + ty < wA); // used to computations and to write in output matrix

  // Loop over all the sub-matrices of A and B required to compute the block sub-matrix
  for (int a = begin_A, b = begin_B; a <= end_A; a += step_A, b += step_B) {

    // Load the matrices from device memory to shared memory; each thread loads one element of each matrix
    if (a/pA + ty < dim){
      shared_A[ty][tx] = (cond0)? A[a + pA * ty + tx] : 0;
      shared_B[ty][tx] = (cond1)? B[b + pB * ty + tx] : 0;
    }
    else{
      shared_A[ty][tx] = 0;
      shared_B[ty][tx] = 0;
    }

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Compute the difference between the two matrixes; each thread computes one element of the block sub-matrix
    if (cond2 && cond1){
      for (int k = 0; k < BLOCK_DIM; ++k){
        tmp = shared_A[k][ty] - shared_B[k][tx];
        ssd += tmp*tmp;
      }
    }

    // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory; each thread writes one element
  if (cond2 && cond1)
    AB[ (begin_A + ty) * pB + begin_B + tx ] = ssd;
}

__global__ void cuInsertionSort(float *dist, int dist_pitch, int *ind, int ind_pitch, int width, int height, int k){

  // Variables
  int l, i, j;
  float *p_dist;
  int   *p_ind;
  float curr_dist, max_dist;
  int   curr_row,  max_row;
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

  if (xIndex<width){

    // Pointer shift, initialization, and max value
    p_dist   = dist + xIndex;
    p_ind    = ind  + xIndex;
    max_dist = p_dist[0];
    p_ind[0] = 1;

    // Part 1 : sort kth firt elementZ
    for (l=1; l<k; l++){
      curr_row  = l * dist_pitch;
      curr_dist = p_dist[curr_row];
      if (curr_dist<max_dist){
        i=l-1;
        for (int a=0; a<l-1; a++){
          if (p_dist[a*dist_pitch]>curr_dist){
            i=a;
            break;
          }
        }
        for (j=l; j>i; j--){
          p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
          p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
        }
        p_dist[i*dist_pitch] = curr_dist;
        p_ind[i*ind_pitch]   = l+1;
      }
      else{
        p_ind[l*ind_pitch] = l+1;
      }
      max_dist = p_dist[curr_row];
    }

    // Part 2 : insert element in the k-th first lines
    max_row = (k-1)*dist_pitch;
    for (l=k; l<height; l++){
      curr_dist = p_dist[l*dist_pitch];
      if (curr_dist<max_dist){
        i=k-1;
        for (int a=0; a<k-1; a++){
          if (p_dist[a*dist_pitch]>curr_dist){
            i=a;
            break;
          }
        }
        for (j=k-1; j>i; j--){
          p_dist[j*dist_pitch] = p_dist[(j-1)*dist_pitch];
          p_ind[j*ind_pitch]   = p_ind[(j-1)*ind_pitch];
        }
        p_dist[i*dist_pitch] = curr_dist;
        p_ind[i*ind_pitch]   = l+1;
        max_dist             = p_dist[max_row];
      }
    }
  }
}


__global__ void cuParallelSqrt(float *dist, int width, int pitch, int k){
  unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
  if (xIndex<width && yIndex<k)
    dist[yIndex*pitch + xIndex] = sqrt(dist[yIndex*pitch + xIndex]);
}



void printErrorMessage(hipError_t error, int memorySize){
  printf("==================================================\n");
  printf("MEMORY ALLOCATION ERROR  : %s\n", hipGetErrorString(error));
  printf("Whished allocated memory : %d\n", memorySize);
  printf("==================================================\n");
}



void knn_brute_force(float* ref_host, int ref_width, float* query_host, int query_width, int height, int k, float* dist_host, int* ind_host){

  unsigned int size_of_float = sizeof(float);
  unsigned int size_of_int   = sizeof(int);

  // Variables
  float        *query_dev;
  float        *ref_dev;
  float        *dist_dev;
  int          *ind_dev;
  hipArray    *ref_array;
  hipError_t  result;
  size_t       query_pitch;
  size_t       query_pitch_in_bytes;
  size_t       ref_pitch;
  size_t       ref_pitch_in_bytes;
  size_t       ind_pitch;
  size_t       ind_pitch_in_bytes;
  size_t       max_nb_query_traited;
  size_t       actual_nb_query_width;
  size_t       memory_total;
  size_t       memory_free;


  unsigned int use_texture = ( ref_width*size_of_float<=MAX_TEXTURE_WIDTH_IN_BYTES && height*size_of_float<=MAX_TEXTURE_HEIGHT_IN_BYTES );

  hipInit(0);

  hipCtx_t cuContext;
  hipDevice_t  cuDevice=0;
  hipCtxCreate(&cuContext, 0, cuDevice);
  hipMemGetInfo(&memory_free, &memory_total);
  cuCtxDetach (cuContext);
  max_nb_query_traited = ( memory_free * MAX_PART_OF_FREE_MEMORY_USED - size_of_float * ref_width*height ) / ( size_of_float * (height + ref_width) + size_of_int * k);
  max_nb_query_traited = min( query_width, (max_nb_query_traited / 16) * 16 );

  result = hipMallocPitch( (void **) &query_dev, &query_pitch_in_bytes, max_nb_query_traited * size_of_float, height + ref_width);
  if (result){
    printErrorMessage(result, max_nb_query_traited*size_of_float*(height+ref_width));
    return;
  }
  query_pitch = query_pitch_in_bytes/size_of_float;
  dist_dev    = query_dev + height * query_pitch;

  result = hipMallocPitch( (void **) &ind_dev, &ind_pitch_in_bytes, max_nb_query_traited * size_of_int, k);
  if (result){
    hipFree(query_dev);
    printErrorMessage(result, max_nb_query_traited*size_of_int*k);
    return;
  }
  ind_pitch = ind_pitch_in_bytes/size_of_int;

  if (use_texture){

    hipChannelFormatDesc channelDescA = hipCreateChannelDesc<float>();
    result = hipMallocArray( &ref_array, &channelDescA, ref_width, height );
    if (result){
      printf("ref_array\n");
      printErrorMessage(result, ref_width*height*size_of_float);
      hipFree(ind_dev);
      hipFree(query_dev);
      return;
    }
    hipMemcpyToArray( ref_array, 0, 0, ref_host, ref_width * height * size_of_float, hipMemcpyHostToDevice );

    texA.addressMode[0] = hipAddressModeClamp;
    texA.addressMode[1] = hipAddressModeClamp;
    texA.filterMode     = hipFilterModePoint;
    texA.normalized     = 0;
    hipBindTextureToArray(texA, ref_array);

  }
  else{

    // Allocation of global memory
    result = hipMallocPitch( (void **) &ref_dev, &ref_pitch_in_bytes, ref_width * size_of_float, height);
    if (result){
      printErrorMessage(result,  ref_width*size_of_float*height);
      hipFree(ind_dev);
      hipFree(query_dev);
      return;
    }
    ref_pitch = ref_pitch_in_bytes/size_of_float;
    hipMemcpy2D(ref_dev, ref_pitch_in_bytes, ref_host, ref_width*size_of_float,  ref_width*size_of_float, height, hipMemcpyHostToDevice);
  }

  // Split queries to fit in GPU memory
  for (int i=0; i<query_width; i+=max_nb_query_traited){

    // Number of query points considered
    actual_nb_query_width = min( max_nb_query_traited, query_width-i );

    // Copy of part of query actually being treated
    hipMemcpy2D(query_dev, query_pitch_in_bytes, &query_host[i], query_width*size_of_float, actual_nb_query_width*size_of_float, height, hipMemcpyHostToDevice);

    // Grids ans threads
    dim3 g_16x16(actual_nb_query_width/16, ref_width/16, 1);
    dim3 t_16x16(16, 16, 1);
    if (actual_nb_query_width%16 != 0){
     g_16x16.x += 1;
    }
    if (ref_width  %16 != 0){
     g_16x16.y += 1;
    }
    //
    dim3 g_256x1(actual_nb_query_width/256, 1, 1);
    dim3 t_256x1(256, 1, 1);
    if (actual_nb_query_width%256 != 0){
     g_256x1.x += 1;
    }
    //
    dim3 g_k_16x16(actual_nb_query_width/16, k/16, 1);
    dim3 t_k_16x16(16, 16, 1);
    if (actual_nb_query_width%16 != 0){
     g_k_16x16.x += 1;
    }
    if (k  %16 != 0){
     g_k_16x16.y += 1;
    }

    // Kernel 1: Compute all the distances
    if (use_texture){
      cuComputeDistanceTexture<<<g_16x16,t_16x16>>>(ref_width, query_dev, actual_nb_query_width, query_pitch, height, dist_dev);
    }
    else{
      cuComputeDistanceGlobal<<<g_16x16,t_16x16>>>(ref_dev, ref_width, ref_pitch, query_dev, actual_nb_query_width, query_pitch, height, dist_dev);
    }

    // Kernel 2: Sort each column
    cuInsertionSort<<<g_256x1,t_256x1>>>(dist_dev, query_pitch, ind_dev, ind_pitch, actual_nb_query_width, ref_width, k);

    // Kernel 3: Compute square root of k first elements
    cuParallelSqrt<<<g_k_16x16,t_k_16x16>>>(dist_dev, query_width, query_pitch, k);

    // Memory copy of output from device to host
    hipMemcpy2D(&dist_host[i], query_width*size_of_float, dist_dev, query_pitch_in_bytes, actual_nb_query_width*size_of_float, k, hipMemcpyDeviceToHost);
    hipMemcpy2D(&ind_host[i],  query_width*size_of_int,   ind_dev,  ind_pitch_in_bytes,   actual_nb_query_width*size_of_int,   k, hipMemcpyDeviceToHost);
  }

  // Free memory
  if (use_texture){
    hipFreeArray(ref_array);
  }
  else{
    hipFree(ref_dev);
    hipFree(ind_dev);
    hipFree(query_dev);
  }
}
