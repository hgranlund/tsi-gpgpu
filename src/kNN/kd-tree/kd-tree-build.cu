#include "hip/hip_runtime.h"
#include "kd-tree-build.cuh"
#include "multiple-radix-select.cuh"
#include "quick-select.cuh"
#include "radix-select.cuh"

#include "stdio.h"
#include "point.h"

#include "hip/hip_runtime_api.h"

int nextPowerOf2_(int x)
{
    --x;
    x |= x >> 1;
    x |= x >> 2;
    x |= x >> 4;
    x |= x >> 8;
    x |= x >> 16;
    return ++x;
}

void UpDim(int &dim)
{
    dim = (dim + 1) % 3;
}

void getThreadAndBlockCountForBuild(int n, int &blocks, int &threads)
{
    threads = min(nextPowerOf2_(n), 512);
    blocks = n / threads;
    blocks = max(1, blocks);
    blocks = min(MAX_BLOCK_DIM_SIZE, blocks);
    // printf("block = %d, threads = %d, n = %d\n", blocks, threads, n);
}

__device__ void cuCalculateBlockOffsetAndNoOfLists_(int n, int &n_per_block, int &block_offset)
{
    int rest = n % gridDim.x;

    n_per_block = n / gridDim.x;
    block_offset = n_per_block * blockIdx.x;

    if (rest >= gridDim.x - blockIdx.x)
    {
        block_offset += rest - (gridDim.x - blockIdx.x);
        n_per_block++;
    }
}

__device__ void cuPointSwapCondition(struct Point *p, int a, int b, int dim)
{
    struct Point temp_a = p[a], temp_b = p[b];
    if (temp_a.p[dim] > temp_b.p[dim] )
    {
        p[a] = temp_b, p[b] = temp_a;
    }
}

__global__ void balanceLeafs(struct Point *points, int *steps, int p, int dim)
{
    struct Point   *l_points;

    int list_in_block,
        block_offset,
        tid = threadIdx.x,
        step_num,
        n;

    cuCalculateBlockOffsetAndNoOfLists_(p, list_in_block, block_offset);

    steps += block_offset * 2;

    while ( tid < list_in_block)
    {
        step_num =  tid * 2;
        l_points = points + steps[step_num];
        n = steps[step_num + 1] - steps[step_num];
        if (n == 2)
        {
            cuPointSwapCondition(l_points, 0, 1, dim);
        }
        else if (n == 3)
        {
            cuPointSwapCondition(l_points, 0, 1, dim);
            cuPointSwapCondition(l_points, 1, 2, dim);
            cuPointSwapCondition(l_points, 0, 1, dim);
        }
        tid += blockDim.x;
    }
}

int store_locations(struct Node *tree, int lower, int upper, int n)
{
    int r;

    if (lower >= upper)
    {
        return -1;
    }

    r = (int) ((upper - lower) / 2) + lower;

    tree[r].left = store_locations(tree, lower, r, n);
    tree[r].right = store_locations(tree, r + 1, upper, n);

    return r;
}

__device__ __host__
void pointConvert(struct Node &p1, struct Point &p2)
{
    p1.p[0] = p2.p[0], p1.p[1] = p2.p[1], p1.p[2] = p2.p[2];
#if KEEP_POINT_INDEX
    p1.index_orig = p2.index_orig;
#endif
}

__global__
void convertPoints(struct Point *points_small, int n, struct Node *points)
{
    int local_n,
        block_offset,
        tid = threadIdx.x;

    cuCalculateBlockOffsetAndNoOfLists_(n, local_n, block_offset);

    points += block_offset;
    points_small += block_offset;

    while (tid < local_n)
    {
        pointConvert(points[tid], points_small[tid]);
        tid += blockDim.x;
    }
}

void nextStep(int *steps_new, int *steps_old, int n)
{
    int i, midpoint, from, to;
    for (i = 0; i < n / 2; ++i)
    {
        from = steps_old[i * 2];
        to = steps_old[i * 2 + 1];
        midpoint = (to - from) / 2 + from;

        steps_new[i * 4] = from;
        steps_new[i * 4 + 1] = midpoint;
        steps_new[i * 4 + 2] = midpoint + 1;
        steps_new[i * 4 + 3] = to;
    }
}

void swap_pointer(int **a, int **b)
{
    int *swap;
    swap = *a, *a = *b, *b = swap;
}

void singleRadixSelectAndPartition(struct Point *d_points, struct Point *d_swap, int *d_partition, int *h_steps, int p, int  dir)
{
    int nn, offset, j;
    for (j = 0; j < p; j ++)
    {
        offset = h_steps[j * 2];
        nn = h_steps[j * 2 + 1] - offset;
        if (nn > 1)
        {
            radixSelectAndPartition(d_points + offset, d_swap + offset, d_partition + offset, nn, dir);
        }
    }
}

size_t getFreeBytesOnGpu_()
{
    size_t free_byte, total_byte ;
    hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
    return free_byte;
}

size_t getNeededBytesForBuildingKdTree(int n)
{
    int number_of_leafs = (n + 1) / 2;
    return (number_of_leafs * 2 * sizeof(int)) + (2 * n * sizeof(int)) + (2 * n * sizeof(Point));
}

void cuBuildKdTree(struct Point *h_points, int n, int dim, struct Node *tree)
{
    struct Point *d_points, *d_swap;
    struct Node *d_tree;
    int *d_partition,
        block_num, thread_num,
        *d_steps, *h_steps_old, *h_steps_new,
        step,
        i = 0,
        p = 1,
        number_of_leafs = (n + 1) / 2,
        h = (int)ceil(log2((float)n + 1));

    h_steps_new = (int *)malloc(number_of_leafs * 2 * sizeof(int));
    h_steps_old = (int *)malloc(number_of_leafs * 2 * sizeof(int));

    h_steps_new[0] = 0;
    h_steps_old[0] = 0;
    h_steps_old[1] = n;
    h_steps_new[1] = n;

    checkCudaErrors(
        hipMalloc(&d_steps, number_of_leafs * 2 * sizeof(int)));
    checkCudaErrors(
        hipMalloc(&d_partition, n * sizeof(int)));
    checkCudaErrors(
        hipMalloc(&d_points, n * sizeof(Point)));
    checkCudaErrors(
        hipMalloc(&d_swap, n * sizeof(Point)));

    checkCudaErrors(
        hipMemcpy(d_points, h_points, n * sizeof(Point), hipMemcpyHostToDevice));

    radixSelectAndPartition(d_points, d_swap, d_partition, n, dim);

    UpDim(dim);
    i++;
    while (i < (h - 1) )
    {
        nextStep(h_steps_new, h_steps_old, p <<= 1);
        step = h_steps_new[1] - h_steps_new[0];
        checkCudaErrors(
            hipMemcpy(d_steps, h_steps_new, p * 2 * sizeof(int), hipMemcpyHostToDevice));

        if (step >= 9000000)
        {
            singleRadixSelectAndPartition(d_points, d_swap, d_partition, h_steps_new, p, dim);
        }
        else if (step > 3000)
        {
            multiRadixSelectAndPartition(d_points, d_swap, d_partition, d_steps, step, p, dim);
        }
        else if (step > 3)
        {
            quickSelectAndPartition(d_points, d_steps, step, p, dim);
        }
        else
        {
            getThreadAndBlockCountForBuild(n, block_num, thread_num);
            balanceLeafs <<< block_num, thread_num >>> (d_points, d_steps, p, dim);
        }
        swap_pointer(&h_steps_new, &h_steps_old);
        i++;
        UpDim(dim);
    }

    checkCudaErrors(hipFree(d_swap));
    checkCudaErrors(hipFree(d_partition));
    checkCudaErrors(hipFree(d_steps));
    free(h_steps_new);
    free(h_steps_old);

    checkCudaErrors(hipMalloc(&d_tree, n * sizeof(Node)));

    getThreadAndBlockCountForBuild(n, block_num, thread_num);
    convertPoints <<< block_num, thread_num >>> (d_points, n, d_tree);

    checkCudaErrors(hipMemcpy(tree, d_tree, n * sizeof(Node), hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_points));
    checkCudaErrors(hipFree(d_tree));
}

void buildKdTreeStep(struct Point *h_points, int n, int dim, struct Node *tree)
{
    if (n <= 0) return;

    size_t free_bytes, needed_bytes;
    int m = n >> 1;

    free_bytes = getFreeBytesOnGpu_();
    needed_bytes = getNeededBytesForBuildingKdTree(n);

    if (free_bytes > needed_bytes)
    {
        cuBuildKdTree(h_points, n, dim, tree);
    }
    else
    {
        cpuQuickSelect(h_points, n, dim);
        pointConvert(tree[m], h_points[m]);

        UpDim(dim);

        buildKdTreeStep(h_points, m, dim, tree);
        buildKdTreeStep(h_points + m + 1, n - m - 1, dim, tree + m + 1);
    }
}


void buildKdTree(struct Point *h_points, int n, struct Node *tree)
{

    int dim = 0;
    buildKdTreeStep(h_points, n, dim, tree);
    store_locations(tree, 0, n, n);
}
