#include "hip/hip_runtime.h"
#include "kd-tree-build.cuh"
#include "multiple-radix-select.cuh"
#include "quick-select.cuh"
#include "radix-select.cuh"

#include "stdio.h"
#include "point.h"

#include "hip/hip_runtime_api.h"

#define debug 0
#include "common-debug.cuh"


__device__ void cuPointSwapCondition(struct PointS *p, int a, int b, int dim)
{
    struct PointS temp_a = p[a], temp_b = p[b];
    if (temp_a.p[dim] > temp_b.p[dim] )
    {
        p[a] = temp_b, p[b] = temp_a;
    }
}

__global__ void balance_leafs(struct PointS *points, int *steps, int p, int dim)
{
    struct PointS   *l_points;

    int
    list_in_block = p / gridDim.x,
    block_offset = list_in_block * blockIdx.x,
    tid = threadIdx.x,
    rest = p % gridDim.x,
    step_num,
    n;

    if (rest >= gridDim.x - blockIdx.x)
    {
        block_offset += rest - (gridDim.x - blockIdx.x);
        list_in_block++;
    }
    steps += block_offset * 2;
    while ( tid < list_in_block)
    {
        step_num =  tid * 2;
        l_points = points + steps[step_num];
        n = steps[step_num + 1] - steps[step_num];
        if (n == 2)
        {
            cuPointSwapCondition(l_points, 0, 1, dim);
        }
        else if (n == 3)
        {
            cuPointSwapCondition(l_points, 0, 1, dim);
            cuPointSwapCondition(l_points, 1, 2, dim);
            cuPointSwapCondition(l_points, 0, 1, dim);
        }
        tid += blockDim.x;
    }
}

int store_locations(struct Point *tree, int lower, int upper, int n)
{
    int r;

    if (lower >= upper)
    {
        return -1;
    }

    r = (int) ((upper - lower) / 2) + lower;

    tree[r].left = store_locations(tree, lower, r, n);
    tree[r].right = store_locations(tree, r + 1, upper, n);

    return r;
}

__global__
void convertPoints(struct PointS *points_small, int n, struct Point *points)
{
    int
    block_stride = n / gridDim.x,
    block_offset = block_stride * blockIdx.x,
    tid = threadIdx.x,
    rest = n % gridDim.x;
    struct PointS point_s;
    if (rest >= gridDim.x - blockIdx.x)
    {
        block_offset += rest - (gridDim.x - blockIdx.x);
        block_stride++;
    }
    points += block_offset;
    while (tid < block_stride)
    {
        struct Point point;
        point_s = points_small[tid];
        point.p[0] = point_s.p[0];
        point.p[1] = point_s.p[1];
        point.p[2] = point_s.p[2];
        points[tid] = point;
        tid += blockDim.x;
    }
}

void nextStep(int *steps_new, int *steps_old, int n)
{
    int midpoint, from, to;
    for (int i = 0; i < n / 2; ++i)
    {
        from = steps_old[i * 2];
        to = steps_old[i * 2 + 1];
        midpoint = (to - from) / 2 + from;
        steps_new[i * 4] = from;
        steps_new[i * 4 + 1] = midpoint;
        steps_new[i * 4 + 2] = midpoint + 1;
        steps_new[i * 4 + 3] = to;
    }
}

void swap_pointer(int **a, int **b)
{
    int *swap;
    swap = *a, *a = *b, *b = swap;

}

void singleRadixSelectAndPartition(struct PointS *d_points, struct PointS *d_swap, int *d_partition, int *h_steps, int p, int  dir)
{
    int nn, offset, j;
    for (j = 0; j < p; j ++)
    {
        offset = h_steps[j * 2];
        nn = h_steps[j * 2 + 1] - offset;
        if (nn > 1)
        {
            radixSelectAndPartition(d_points + offset, d_swap + offset, d_partition + offset, nn, dir);
        }
    }
}

void build_kd_tree(struct PointS *h_points, int n, struct Point *h_points_out)
{
    struct PointS *d_points, *d_swap;
    struct Point *d_points_out;
    int p, h, i, *d_partition,
        *d_steps, *h_steps_old, *h_steps_new;

    h_steps_new = (int *)malloc(n * 2 * sizeof(int));
    h_steps_old = (int *)malloc(n * 2 * sizeof(int));

    checkCudaErrors(
        hipMalloc(&d_steps, n * 2 * sizeof(int)));

    checkCudaErrors(
        hipMalloc(&d_partition, n * sizeof(int)));

    checkCudaErrors(
        hipMalloc(&d_points, n * sizeof(PointS)));

    checkCudaErrors(
        hipMalloc(&d_swap, n * sizeof(PointS)));

    checkCudaErrors(
        hipMemcpy(d_points, h_points, n * sizeof(PointS), hipMemcpyHostToDevice));

    p = 1;
    i = 0;
    int step;
    h = ceil(log2((float)n + 1));
    h_steps_new[0] = 0;
    h_steps_old[0] = 0;
    h_steps_old[1] = n;
    h_steps_new[1] = n;

    radixSelectAndPartition(d_points, d_swap, d_partition, n, i % 3);
    i++;
    while (i < (h - 1) )
    {
        nextStep(h_steps_new, h_steps_old, p <<= 1);
        step = h_steps_new[1] - h_steps_new[0];
        checkCudaErrors(
            hipMemcpy(d_steps, h_steps_new, p * 2 * sizeof(int), hipMemcpyHostToDevice));

        if (step >= 8388608)
        {
            singleRadixSelectAndPartition(d_points, d_swap, d_partition, h_steps_new, p, i % 3);
        }

        else if (step > 4000)
        {
            multiRadixSelectAndPartition(d_points, d_swap, d_partition, d_steps, step, p, i % 3);
        }
        else if (step > 3)
        {
            quickSelectAndPartition(d_points, d_steps, step, p, i % 3);
        }
        else
        {
            balance_leafs <<< 1, min(n, 512) >>> (d_points, d_steps, p, i % 3);
        }
        swap_pointer(&h_steps_new, &h_steps_old);
        i++;
    }

    checkCudaErrors(hipFree(d_swap));
    checkCudaErrors(hipFree(d_partition));
    checkCudaErrors(hipFree(d_steps));
    free(h_steps_new);
    free(h_steps_old);

    checkCudaErrors(hipMalloc(&d_points_out, n * sizeof(Point)));

    convertPoints <<< max(1, n / 512), 512 >>> (d_points, n, d_points_out);
    checkCudaErrors(hipMemcpy(h_points_out, d_points_out, n * sizeof(Point), hipMemcpyDeviceToHost));
    store_locations(h_points_out, 0, n, n);

    checkCudaErrors(hipFree(d_points));
}


