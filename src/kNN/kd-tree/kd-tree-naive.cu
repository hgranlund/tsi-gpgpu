#include "hip/hip_runtime.h"
#include <kd-tree-naive.cuh>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <point.h>
#include <hip/hip_runtime_api.h>

#define THREADS_PER_BLOCK 1024U
#define MAX_BLOCK_DIM_SIZE 65535U
// #define THREADS_PER_BLOCK 4U
// #define MAX_BLOCK_DIM_SIZE 8U

#include <string.h>

#define debug 0
#include "common-debug.cuh"


__global__ void cuRadixSelectGlobal(Point *data, Point *data_copy, unsigned int m, unsigned int n, int *partition, int dir)
{
  cuRadixSelect(data, data_copy, m, n, partition, dir);
}



__device__ __host__
unsigned int nextPowerOf2(unsigned int x)
{
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

__device__ __host__
bool isPowTwo(unsigned int x)
{
  return ((x&(x-1))==0);
}

__device__ __host__
unsigned int prevPowerOf2(unsigned int n)
{
    if (isPowTwo(n))
    {
        return n;
    }
    n = nextPowerOf2(n);
    return n >>=1;

}



__device__ void cuPointSwap(Point *p, int a, int b){
    Point temp = p[a];
    p[a]=p[b], p[b]=temp;
}

__device__ int cuSumReduce(int *list, int n)
{
  int half = n/2;
  int tid = threadIdx.x;
  while(tid<half && half > 0)
  {
    list[tid] += list[tid+half];
    half = half/2;
}
return list[0];
}

//TODO must be imporved
__device__  void cuAccumulateIndex(int *list, int n)
{
    if (threadIdx.x == 0)
    {
        int sum=0;
        list[n]=list[n-1];
        int temp=0;
        for (int i = 0; i < n; ++i)
        {
            temp = list[i];
            list[i] = sum;
            sum += temp;
        }
        list[n]+=list[n-1];
    }
}


__device__ void cuPartitionSwap(Point *data, Point *swap, unsigned int n, int *partition, int *zero_count, int *one_count, Point median, int dir)
{
    unsigned int
    tid = threadIdx.x,
    is_bigger,
    big,
    less;

    zero_count[threadIdx.x] = 0;
    one_count[threadIdx.x] = 0;

    while(tid < n)
    {
        swap[tid]=data[tid];
        is_bigger = partition[tid]= (bool)(data[tid].p[dir] > median.p[dir]);
        one_count[threadIdx.x] += is_bigger;
        zero_count[threadIdx.x] += !is_bigger;
        tid+=blockDim.x;
    }
    __syncthreads();
    cuAccumulateIndex(zero_count, blockDim.x);
    cuAccumulateIndex(one_count, blockDim.x);
    tid = threadIdx.x;
    __syncthreads();
    less = zero_count[threadIdx.x];
    big = one_count[threadIdx.x];
    while(tid<n)
    {
        if (!partition[tid])
        {
            data[less]=swap[tid];
            less++;
        }else
        {
            data[n-big-1]=swap[tid];
            big++;
        }
        tid+=blockDim.x;
    }
}

__device__ unsigned int cuPartition(Point *data, unsigned int n, int *partition, int *zero_count, int last, unsigned int bit, int dir)
{
    unsigned int
    tid = threadIdx.x,
    is_one,
    radix = (1 << 31-bit);
    zero_count[threadIdx.x] = 0;

    while(tid < n)
    {
        if (partition[tid] == last)
        {
            is_one = partition[tid]= (bool)((*(int*)&(data[tid].p[dir]))&radix);
            zero_count[threadIdx.x] += !is_one;
        }else{
            partition[tid] = 2;
        }
        tid+=blockDim.x;
    }
    return cuSumReduce(zero_count, blockDim.x);
}


__device__ void cuRadixSelect(Point *data, Point *data_copy, unsigned int m, unsigned int n, int *partition, int dir)
{
    __shared__ int one_count[1025];
    __shared__ int zeros_count[1025];
    __shared__ Point median;


    int l=0,
    u = n,
    cut=0,
    bit = 0,
    last = 2,
    tid = threadIdx.x;
    while(tid < n)
    {
        partition[tid] = last;
        tid+=blockDim.x;
    }

    tid = threadIdx.x;
    do {
        __syncthreads();
        cut = cuPartition(data, n, partition, zeros_count, last, bit++, dir);
        if ((l+cut) <= m)
        {
            l +=cut;
            last = 1;
        }
        else
        {
            last = 0;
            u -=u-cut-l;
        }
    }while (((u-l)>1) && (bit<32));

    tid = threadIdx.x;

    __syncthreads();
    while(tid < n)
    {
        if (partition[tid] == last)
        {
            median = data[tid];
            data[tid]=data[0], data[0] = median;
        }
        tid+=blockDim.x;
    }
    __syncthreads();
    cuPartitionSwap(data+1, data_copy, n-1, partition, one_count, zeros_count, median, dir);
    median = data[m];
    data[m]=data[0], data[0] = median;
}

__global__
void cuBalanceBranchLeafs(Point* points, int n, int dir)
{
    int
    step = n/gridDim.x,
    blockOffset = step*blockIdx.x,
    tid = threadIdx.x;
    step=step/2;
    Point point1;
    Point point2;
    points += blockOffset;
    while(tid < step){
        point1 = points[tid*2];
        point2 = points[tid*2+1];
        if (point1.p[dir]>point2.p[dir])
        {
            points[tid*2] = point2;
            points[tid*2+1] = point1;
        }
        tid += blockDim.x;
    }
}

__global__
void cuQuickSelect(Point* points, int n, int p, int *blockOffsets, int dir){

    int pos, i,
    step=n,
    listInBlock = blockOffsets[blockIdx.x+1]-blockOffsets[blockIdx.x],
    right,
    left,
    tid = threadIdx.x,
    m=step/2;
    points += blockOffsets[blockIdx.x]*step;
    points += step * tid;
    float pivot;
    while( tid < listInBlock)
    {
        left = 0;
        right = step - 1;
        while (left < right)
        {
            pivot = points[m].p[dir];
            cuPointSwap(points, m, right);
            for (i = pos = left; i < right; i++)
            {
                if (points[i].p[dir] < pivot)
                {
                    cuPointSwap(points, i, pos);
                    pos++;
                }
            }
            cuPointSwap(points, right, pos);
            if (pos == m) break;
            if (pos < m) left = pos + 1;
            else right = pos - 1;
        }
        tid += blockDim.x;
        points += step * blockDim.x;

    }
}




__global__
void cuBalanceBranch(Point* points, Point* swap, int *partition, int n, int p, int dir){

    int blockoffset, bid;
    bid = blockIdx.x;
    while(bid < p){
        blockoffset = n * bid;
        cuRadixSelect(points+blockoffset, swap+blockoffset, n/2, n, partition+blockoffset, dir);
        bid += gridDim.x;
    }
}
void getThreadAndBlockCount(int n, int p, int &blocks, int &threads)
{
    n = n/p;
    n = prevPowerOf2(n/2);
    blocks = min(MAX_BLOCK_DIM_SIZE, p);
    blocks = max(1, blocks);
    threads = min(THREADS_PER_BLOCK, n);
    threads = max(1, threads);
}
void getThreadAndBlockCountForQuickSelect(int n, int p, int &blocks, int &threads)
{
    threads = 128;
    int step = n/p,
    numberOfLists= n/step;
    blocks = numberOfLists/threads;
    blocks = min(MAX_BLOCK_DIM_SIZE, blocks);
    blocks = max(1, blocks);
}

void build_kd_tree(Point *h_points, int n)
{


    Point *d_points, *d_swap;
    int p, h, i, j, numBlocks, numThreads, *h_blockOffsets, *d_blockOffsets;
    int *d_partition;

    checkCudaErrors(
        hipMalloc(&d_partition, n*sizeof(int)));

    checkCudaErrors(
        hipMalloc(&d_points, n*sizeof(Point)));

    checkCudaErrors(
        hipMalloc(&d_swap, n*sizeof(Point)));

    checkCudaErrors(
        hipMemcpy(d_points, h_points, n*sizeof(Point), hipMemcpyHostToDevice));

    h = ceil(log2((float)n + 1) - 1);
    p = 1;
    for (i = 0; i < h-4; i++)
    {
        getThreadAndBlockCount(n, p, numBlocks, numThreads);
        debugf("n = %d, p = %d, numblosck = %d, numThread =%d\n", n/p, p, numBlocks, numThreads );
        cuBalanceBranch<<<numBlocks,numThreads>>>(d_points, d_swap, d_partition, n/p, p, i%3);
        p <<=1;
    }
    for (int i = max(h-4, 0) ; i < h-1; ++i)
    {

        getThreadAndBlockCountForQuickSelect(n, p, numBlocks, numThreads);
        h_blockOffsets = (int*) malloc((numBlocks+1)*sizeof(int));
        h_blockOffsets[numBlocks]=p;
        h_blockOffsets[0]=0;
       for (j = 1; j < numBlocks; ++j)
        {
            h_blockOffsets[j]=p/numBlocks * j;
        }
        int rest = p % numBlocks;
        for (j = n-1; j >= n-(p % numBlocks); --j)
        {
            h_blockOffsets[j]+=rest;
            rest--;
        }


        checkCudaErrors(
            hipMalloc((void **)&d_blockOffsets, (numBlocks+1)*sizeof(int)));
        checkCudaErrors(
            hipMemcpy(d_blockOffsets, h_blockOffsets, (numBlocks+1)*sizeof(int), hipMemcpyHostToDevice));
        debugf("n = %d, p = %d, numblock = %d, numThread =%d, rest = %d, i = %d, h=%d\n", n/p, p, numBlocks, numThreads,p % numBlocks, i ,h );
        cuQuickSelect<<<numBlocks,numThreads>>>(d_points, n/p, p, d_blockOffsets, i%3);
        p <<=1;
        checkCudaErrors(
            hipFree(d_blockOffsets));

            // checkCudaErrors(
        // hipMemcpy(h_points, d_points, n*sizeof(Point), hipMemcpyDeviceToHost));

        // h_printPointsArray(h_points, n, "after step");
    }

    numThreads = min(n, THREADS_PER_BLOCK/2);
    numBlocks = n/numThreads;
    numBlocks = min(numBlocks, 65536);
    debugf("n = %d, p = %d, numblosck = %d, numThread =%d\n", n/p, p, numBlocks, numThreads );
    cuBalanceBranchLeafs<<<numBlocks, numThreads>>>(d_points, n, (h-1)%3);

    checkCudaErrors(
        hipMemcpy(h_points, d_points, n*sizeof(Point), hipMemcpyDeviceToHost));


    checkCudaErrors(hipFree(d_points));
    checkCudaErrors(hipFree(d_swap));
    checkCudaErrors(hipFree(d_partition));
    free(h_blockOffsets);
}


