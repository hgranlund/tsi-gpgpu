#include "hip/hip_runtime.h"
#include "kd-tree-naive.cuh"
#include "multiple-radix-select.cuh"
#include "quick-select.cuh"
#include "radix-select.cuh"

#include "stdio.h"
#include "point.h"

#include "hip/hip_runtime_api.h"

#define debug 0
#include "common-debug.cuh"


__global__
void cuBalanceBranchLeafs(Point *points, int n, int dir)
{
    int
    step = n / gridDim.x,
    blockOffset = step * blockIdx.x,
    tid = threadIdx.x;
    step = step >> 1;           // same as n / 2;
    Point point1;
    Point point2;
    points += blockOffset;
    while (tid < step)
    {
        point1 = points[tid * 2];
        point2 = points[tid * 2 + 1];
        if (point1.p[dir] > point2.p[dir])
        {
            points[tid * 2] = point2;
            points[tid * 2 + 1] = point1;
        }
        tid += blockDim.x;
    }
}

void nextStep(int *steps_new, int *steps_old, int n)
{
    int midpoint;
    for (int i = 0; i < n / 2; ++i)
    {
        midpoint = steps_old[i * 2 + 1] / steps_old[i * 2];
        steps_new[i * 4] = steps_old[i * 2];
        steps_new[i * 4 + 1] = steps_old[midpoint];
        steps_new[i * 4 + 2] = steps_old[midpoint + 1];
        steps_new[i * 4 + 3] = steps_old[i * 2 + 1];
    }
}


void build_kd_tree(Point *h_points, int n)
{


    Point *d_points, *d_swap;
    int p, i, j, numBlocks, numThreads, step;
    int *d_partition , *d_steps, *h_steps_old, *h_steps_new;

    h_steps_new = (int *)malloc(n * sizeof(int));
    h_steps_old = (int *)malloc(n * sizeof(int));

    checkCudaErrors(
        hipMalloc(&d_steps, n * sizeof(int)));

    checkCudaErrors(
        hipMalloc(&d_partition, n * sizeof(int)));

    checkCudaErrors(
        hipMalloc(&d_points, n * sizeof(Point)));

    checkCudaErrors(
        hipMalloc(&d_swap, n * sizeof(Point)));

    checkCudaErrors(
        hipMemcpy(d_points, h_points, n * sizeof(Point), hipMemcpyHostToDevice));

    p = 1;
    step = n / p;
    i = 0;
    h_steps_new[0] = 0;
    h_steps_old[0] = 0;
    h_steps_old[1] = n;
    h_steps_new[1] = n;
    while (step >= 8388608 && p <= 2)
    {
        int nn, offset;
        for (j = 0; j < p; j ++)
        {
            offset = h_steps_new[j * 2];
            nn = h_steps_new[j * 2 + 1] - offset;
            radixSelectAndPartition(d_points + offset, d_swap + offset, d_partition + offset, nn, i % 3);
        }
        nextStep(h_steps_new, h_steps_old, p <<= 1);
        // p <<= 1;
        i++;
    }
    while (step > 256)
    {
        // nextStep(h_steps_new, h_steps_old, p <<= 1);
        // checkCudaErrors(
        // hipMemcpy(d_steps, h_steps_new, p * sizeof(int), hipMemcpyHostToDevice));
        multiRadixSelectAndPartition(d_points, d_swap, d_partition, step, p, i % 3);
        // p <<= 1;
        step = n / p;
        i++;
    }
    while (step > 2)
    {
        quickSelectAndPartition(d_points, step, p, i % 3);
        p <<= 1;
        step = n / p;
        i++;
    }

    numThreads = min(n, THREADS_PER_BLOCK / 2);
    numBlocks = n / numThreads;
    numBlocks = min(numBlocks, MAX_BLOCK_DIM_SIZE);
    debugf("n = %d, p = %d, numblosck = %d, numThread =%d\n", n / p, p, numBlocks, numThreads );
    cuBalanceBranchLeafs <<< numBlocks, numThreads>>>(d_points, n, i % 3);

    checkCudaErrors(
        hipMemcpy(h_points, d_points, n * sizeof(Point), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_points));
    checkCudaErrors(hipFree(d_swap));
    checkCudaErrors(hipFree(d_partition));
}


