#include "hip/hip_runtime.h"
#include <kd-tree-naive.cuh>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <point.h>
#include <hip/hip_runtime_api.h>

#define THREADS_PER_BLOCK 1024U
#define MAX_BLOCK_DIM_SIZE 65535U
// #define THREADS_PER_BLOCK 4U
// #define MAX_BLOCK_DIM_SIZE 8U

#include <string.h>

#define debug 0
#include "common-debug.cuh"


__global__ void cuRadixSelectGlobal(Point *data, Point *data_copy, unsigned int m, unsigned int n, int *partition, int dir)
{
  cuRadixSelect(data, data_copy, m, n, partition, dir);
}



__device__ __host__
unsigned int nextPowerOf2(unsigned int x)
{
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

__device__ __host__
bool isPowTwo(unsigned int x)
{
  return ((x&(x-1))==0);
}

__device__ __host__
unsigned int prevPowerOf2(unsigned int n)
{
    if (isPowTwo(n))
    {
        return n;
    }
    n = nextPowerOf2(n);
    return n >>=1;

}



__device__ void cuPointSwap(Point *p, int a, int b){
    Point temp = p[a];
    p[a]=p[b], p[b]=temp;
}

__device__ int cuSumReduce(int *list, int n)
{
  int half = n/2;
  int tid = threadIdx.x;
  while(tid<half && half > 0)
  {
    list[tid] += list[tid+half];
    half = half/2;
}
return list[0];
}

//TODO must be imporved
__device__  void cuAccumulateIndex(int *list, int n)
{
    if (threadIdx.x == 0)
    {
        int sum=0;
        list[n]=list[n-1];
        int temp=0;
        for (int i = 0; i < n; ++i)
        {
            temp = list[i];
            list[i] = sum;
            sum += temp;
        }
        list[n]+=list[n-1];
    }
}


__device__ void cuPartitionSwap(Point *data, Point *swap, unsigned int n, int *partition, int *zero_count, int *one_count, Point median, int dir)
{
    unsigned int
    tid = threadIdx.x,
    is_bigger,
    big,
    less;

    zero_count[threadIdx.x] = 0;
    one_count[threadIdx.x] = 0;

    while(tid < n)
    {
        swap[tid]=data[tid];
        is_bigger = partition[tid]= (bool)(data[tid].p[dir] > median.p[dir]);
        one_count[threadIdx.x] += is_bigger;
        zero_count[threadIdx.x] += !is_bigger;
        tid+=blockDim.x;
    }
    __syncthreads();
    cuAccumulateIndex(zero_count, blockDim.x);
    cuAccumulateIndex(one_count, blockDim.x);
    tid = threadIdx.x;
    __syncthreads();
    less = zero_count[threadIdx.x];
    big = one_count[threadIdx.x];
    while(tid<n)
    {
        if (!partition[tid])
        {
            data[less]=swap[tid];
            less++;
        }else
        {
            data[n-big-1]=swap[tid];
            big++;
        }
        tid+=blockDim.x;
    }
}

__device__ unsigned int cuPartition(Point *data, unsigned int n, int *partition, int *zero_count, int last, unsigned int bit, int dir)
{
    unsigned int
    tid = threadIdx.x,
    is_one,
    radix = (1 << 31-bit);
    zero_count[threadIdx.x] = 0;

    while(tid < n)
    {
        if (partition[tid] == last)
        {
            is_one = partition[tid]= (bool)((*(int*)&(data[tid].p[dir]))&radix);
            zero_count[threadIdx.x] += !is_one;
        }else{
            partition[tid] = 2;
        }
        tid+=blockDim.x;
    }
    return cuSumReduce(zero_count, blockDim.x);
}


__device__ void cuRadixSelect(Point *data, Point *data_copy, unsigned int m, unsigned int n, int *partition, int dir)
{
    __shared__ int one_count[1025];
    __shared__ int zeros_count[1025];
    __shared__ Point median;


    int l=0,
    u = n,
    cut=0,
    bit = 0,
    last = 2,
    tid = threadIdx.x;
    while(tid < n)
    {
        partition[tid] = last;
        tid+=blockDim.x;
    }

    tid = threadIdx.x;
    do {
        __syncthreads();
        cut = cuPartition(data, n, partition, zeros_count, last, bit++, dir);
        if ((l+cut) <= m)
        {
            l +=cut;
            last = 1;
        }
        else
        {
            last = 0;
            u -=u-cut-l;
        }
    }while (((u-l)>1) && (bit<32));

    tid = threadIdx.x;

    __syncthreads();
    while(tid < n)
    {
        if (partition[tid] == last)
        {
            median = data[tid];
            data[tid]=data[0], data[0] = median;
        }
        tid+=blockDim.x;
    }
    __syncthreads();
    cuPartitionSwap(data+1, data_copy, n-1, partition, one_count, zeros_count, median, dir);
    median = data[m];
    data[m]=data[0], data[0] = median;
}

__global__
void cuBalanceBranchLeafs(Point* points, int n, int dir)
{
    int
    step = n/gridDim.x,
    blockOffset = step*blockIdx.x,
    tid = threadIdx.x;
    step=step/2;
    Point point1;
    Point point2;
    points += blockOffset;
    while(tid < step){
        point1 = points[tid*2];
        point2 = points[tid*2+1];
        if (point1.p[dir]>point2.p[dir])
        {
            points[tid*2] = point2;
            points[tid*2+1] = point1;
        }
        tid += blockDim.x;
    }
}

__global__
void cuQuickSelect(Point* points, int n, int p, int *blockOffsets, int dir){

    int pos, i,
    right = n/p - 1,
    step=n/p,
    listInBlock = blockOffsets[blockIdx.x+1]-blockOffsets[blockIdx.x],
    left = 0,
    tid = threadIdx.x,
    m=step/2;
    Point s_points[32];
    points += blockOffsets[blockIdx.x]*step;
    points += step * tid;
    float pivot;
    while( tid < listInBlock)
    {
        for (i = 0; i < step; ++i)
        {
            s_points[i]=points[i];
        }
        while (left < right)
        {
            pivot = s_points[m].p[dir];
            cuPointSwap(s_points, m, right);
            for (i = pos = left; i < right; i++)
            {
                if (s_points[i].p[dir] < pivot)
                {
                    cuPointSwap(s_points, i, pos);
                    pos++;
                }
            }
            cuPointSwap(s_points, right, pos);
            if (pos == m) break;
            if (pos < m) left = pos + 1;
            else right = pos - 1;
        }
        for (i = 0; i < step; ++i)
        {
            points[i]=s_points[i];
        }
        tid += blockDim.x;
        points += step * blockDim.x;
    }
    free(s_points);
}



// float quick_select(int k, struct Point *x, int lower, int upper, int dir)
// {


__global__
void cuBalanceBranch(Point* points, Point* swap, int *partition, int n, int p, int dir){

    int blockoffset, bid;
    bid = blockIdx.x;
    while(bid < p){
        blockoffset = n * bid;
        cuRadixSelect(points+blockoffset, swap+blockoffset, n/2, n, partition+blockoffset, dir);
        bid += gridDim.x;
    }
}
void getThreadAndBlockCount(int n, int p, int &blocks, int &threads)
{
    n = n/p;
    n = prevPowerOf2(n/2);
    blocks = min(MAX_BLOCK_DIM_SIZE, p);
    blocks = max(1, blocks);
    threads = min(THREADS_PER_BLOCK, n);
    threads = max(1, threads);
}

void build_kd_tree(Point *h_points, int n)
{


    Point *d_points, *d_swap;
    int p, h, i, numBlocks, numThreads;
    int *d_partition;

    checkCudaErrors(
        hipMalloc(&d_partition, n*sizeof(int)));

    checkCudaErrors(
        hipMalloc(&d_points, n*sizeof(Point)));

    checkCudaErrors(
        hipMalloc(&d_swap, n*sizeof(Point)));

    checkCudaErrors(
        hipMemcpy(d_points, h_points, n*sizeof(Point), hipMemcpyHostToDevice));

    h = ceil(log2((float)n + 1) - 1);
    p = 1;
    for (i = 0; i < h-1; i++)
    {
        getThreadAndBlockCount(n, p, numBlocks, numThreads);
        debugf("n = %d, p = %d, numblosck = %d, numThread =%d\n", n/p, p, numBlocks, numThreads );
        cuBalanceBranch<<<numBlocks,numThreads>>>(d_points, d_swap, d_partition, n/p, p, i%3);
        p <<=1;
    }

    numThreads = min(n, THREADS_PER_BLOCK/2);
    numBlocks = n/numThreads;
    numBlocks = min(numBlocks, 65536);
    debugf("n = %d, p = %d, numblosck = %d, numThread =%d\n", n/p, p, numBlocks, numThreads );
    cuBalanceBranchLeafs<<<numBlocks, numThreads>>>(d_points, n, (h-1)%3);

    checkCudaErrors(
        hipMemcpy(h_points, d_points, n*sizeof(Point), hipMemcpyDeviceToHost));


    checkCudaErrors(hipFree(d_points));
    checkCudaErrors(hipFree(d_swap));
    checkCudaErrors(hipFree(d_partition));
}


