#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include <float.h>

#include <hip/hip_runtime_api.h>
#include <kd-search.cuh>

__device__ __host__
float cuDist(struct Point qp, struct Node point)
{
    float dx = qp.p[0] - point.p[0],
          dy = qp.p[1] - point.p[1],
          dz = qp.p[2] - point.p[2];

    return (dx * dx) + (dy * dy) + (dz * dz);
}

__device__ __host__
void cuInitStack(struct SPoint **stack)
{
    struct SPoint temp;
    temp.index = -1;
    temp.dim = -1;
    cuPush(stack, temp);
}

__device__ __host__
bool cuIsEmpty(struct SPoint *stack)
{
    return cuPeek(stack).index == -1;
}

__device__ __host__
void cuPush(struct SPoint **stack, struct SPoint value)
{
    *((*stack)++) = value;
}

__device__ __host__
struct SPoint cuPop(struct SPoint **stack)
{
    return *(--(*stack));
}

__device__ __host__
struct SPoint cuPeek(struct SPoint *stack)
{
    return *(stack - 1);
}


__device__ __host__
void cuInitKStack(struct KPoint **k_stack, int n)
{
    (*k_stack)[0].dist = -1;
    (*k_stack)++;
    for (int i = 0; i < n; ++i)
    {
        (*k_stack)[i].dist = FLT_MAX;
    }
}

__device__ __host__
void cuInsert(struct KPoint *k_stack, struct KPoint k_point, int n)
{
    int i = n - 1;
    KPoint swap;
    k_stack[n - 1].index = k_point.index;
    k_stack[n - 1].dist = k_point.dist;

    while (k_stack[i].dist < k_stack[i - 1].dist)
    {
        swap = k_stack[i], k_stack[i] = k_stack[i - 1], k_stack[i - 1] = swap;
        i--;
    }
}

__device__ __host__
struct KPoint cuLook(struct KPoint *k_stack, int n)
{
    return k_stack[n - 1];
}

__device__ __host__
void cuUpDim(int *dim)
{
    *dim = (*dim + 1) % 3;
}

__device__ __host__
void cuChildren(struct Point qp, struct Node current, float dx, int &target, int &other)
{
    if (dx > 0)
    {
        other = current.right;
        target = current.left;
    }
    else
    {
        other = current.left;
        target = current.right;
    }
}

__device__ __host__
void cuKNN(struct Point qp, struct Node *tree, int n, int k, int *result,
           struct SPoint *stack_ptr, struct KPoint *k_stack_ptr)
{
    int  dim = 2, target;
    float current_dist, dx, dx2;

    struct Node current_point;
    struct SPoint *stack = stack_ptr,
                           current;
    struct KPoint *k_stack = k_stack_ptr,
                           worst_best;

    current.index = n / 2;
    worst_best.dist = FLT_MAX;

    cuInitStack(&stack);
    cuInitKStack(&k_stack, k);

    while (!cuIsEmpty(stack) || current.index != -1)
{
        if (current.index == -1 && !cuIsEmpty(stack))
        {
            current = cuPop(&stack);
            dim = current.dim;

            dx = current.dx;
            dx2 = dx * dx;

            current.index = (dx2 < worst_best.dist) ? current.other : -1;
        }
        else
        {
            current_point = tree[current.index];

            current_dist = cuDist(qp, current_point);
            if (worst_best.dist > current_dist)
            {
                worst_best.dist = current_dist;
                worst_best.index = current.index;
                cuInsert(k_stack, worst_best, k);
                worst_best = cuLook(k_stack, k);
            }

            cuUpDim(&dim);
            current.dim = dim;
            current.dx = current_point.p[dim] - qp.p[dim];
            cuChildren(qp, current_point, current.dx, target, current.other);
            cuPush(&stack, current);

            current.index = target;
        }
    }

    for (int i = 0; i < k; ++i)
    {
        result[i] = k_stack[i].index;
    }
}

__device__ void cuCalculateBlockOffsetAndNoOfQueries(int n, int &n_per_block, int &block_offset)
{
    int rest = n % gridDim.x;
    n_per_block = n / gridDim.x;
    block_offset = n_per_block * blockIdx.x;

    if (rest >= gridDim.x - blockIdx.x)
    {
        block_offset += rest - (gridDim.x - blockIdx.x);
        n_per_block++;
    }
}

template <int max_k> __global__
void dQueryAll(struct Point *query_points, struct Node *tree, int n_qp, int n_tree, int k, int *result)
{
    int tid = threadIdx.x,
        block_step,
        block_offset;

    struct KPoint *k_stack_ptr = (struct KPoint *) malloc((k + 1) * sizeof(KPoint));
    struct SPoint s_stack_ptr[max_k * THREADS_PER_BLOCK_SEARCH];
    struct SPoint *s_stack = s_stack_ptr + (threadIdx.x * max_k);

    cuCalculateBlockOffsetAndNoOfQueries(n_qp, block_step, block_offset);

    query_points += block_offset;
    result += block_offset * k;

    while (tid < block_step)
    {
        cuKNN(query_points[tid], tree, n_tree, k, result + (tid * k), s_stack, k_stack_ptr);
        tid += blockDim.x;
    }

    free(k_stack_ptr);
}

void getThreadAndBlockCountForQueryAll(int n, int &blocks, int &threads)
{
    threads = THREADS_PER_BLOCK_SEARCH;
    blocks = n / threads;
    blocks = min(MAX_BLOCK_DIM_SIZE, blocks);
    blocks = max(1, blocks);
    // printf("blocks = %d, threads = %d, n= %d\n", blocks, threads, n);
}

void cuQueryAll(struct Point *h_query_points, struct Node *h_tree, int n_qp, int n_tree, int k, int *h_result)
{
    int *d_result, numBlocks, numThreads;
    struct Node *d_tree;
    struct Point *d_query_points;

    checkCudaErrors(hipMalloc(&d_result, n_qp * k  * sizeof(int)));
    checkCudaErrors(hipMalloc(&d_query_points, n_qp * sizeof(Point)));
    checkCudaErrors(hipMalloc(&d_tree, n_tree * sizeof(Node)));

    checkCudaErrors(hipMemcpy(d_query_points, h_query_points, n_qp * sizeof(Point), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_tree, h_tree, n_tree * sizeof(Node), hipMemcpyHostToDevice));

    getThreadAndBlockCountForQueryAll(n_qp, numBlocks, numThreads);

    dQueryAll<20> <<< numBlocks, numThreads>>>(d_query_points, d_tree, n_qp, n_tree, k, d_result);

    checkCudaErrors(hipMemcpy(h_result, d_result, n_qp * k * sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_tree));
    checkCudaErrors(hipFree(d_query_points));
    checkCudaErrors(hipFree(d_result));
}
