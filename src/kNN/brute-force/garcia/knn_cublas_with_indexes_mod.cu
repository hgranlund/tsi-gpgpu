#include "hip/hip_runtime.h"



// Includes
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "hipblas.h"
#include <time.h>


// Constants used by the program
#define MAX_PITCH_VALUE_IN_BYTES       262144
#define MAX_TEXTURE_WIDTH_IN_BYTES     65536
#define MAX_TEXTURE_HEIGHT_IN_BYTES    32768
#define MAX_PART_OF_FREE_MEMORY_USED   0.9
#define BLOCK_DIM                      16

#ifndef max
#define max(a,b) (((a) (b)) ? (a) : (b))
#define min(a,b) (((a) < (b)) ? (a) : (b))
#endif

__global__ void cuComputeNorm(float *mat, int width, int pitch, int height, float *norm)
{
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (xIndex < width)
    {
        float val, sum = 0;
        int i;
        for (i = 0; i < height; i++)
        {
            val  = mat[i * pitch + xIndex];
            sum += val * val;
        }
        norm[xIndex] = sum;
    }
}


__global__ void cuAddRNorm(float *dist, int width, int pitch, int height, float *vec)
{
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int xIndex = blockIdx.x * blockDim.x + tx;
    unsigned int yIndex = blockIdx.y * blockDim.y + ty;
    __shared__ float shared_vec[16];
    if (tx == 0 && yIndex < height)
        shared_vec[ty] = vec[yIndex];
    __syncthreads();
    if (xIndex < width && yIndex < height)
        dist[yIndex * pitch + xIndex] += shared_vec[ty];
}



__global__ void cuInsertionSort(float *dist, int dist_pitch, int *ind, int ind_pitch, int width, int height, int k)
{

    // Variables
    int l, i, j;
    float *p_dist;
    int   *p_ind;
    float curr_dist, max_dist;
    int   curr_row,  max_row;
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (xIndex < width)
    {

        // struct Pointer shift, initialization, and max value
        p_dist   = dist + xIndex;
        p_ind    = ind  + xIndex;
        max_dist = p_dist[0];
        p_ind[0] = 1;

        // Part 1 : sort kth firt elementZ
        for (l = 1; l < k; l++)
        {
            curr_row  = l * dist_pitch;
            curr_dist = p_dist[curr_row];
            if (curr_dist < max_dist)
            {
                i = l - 1;
                for (int a = 0; a < l - 1; a++)
                {
                    if (p_dist[a * dist_pitch] > curr_dist)
                    {
                        i = a;
                        break;
                    }
                }
                for (j = l; j > i; j--)
                {
                    p_dist[j * dist_pitch] = p_dist[(j - 1) * dist_pitch];
                    p_ind[j * ind_pitch]   = p_ind[(j - 1) * ind_pitch];
                }
                p_dist[i * dist_pitch] = curr_dist;
                p_ind[i * ind_pitch]   = l + 1;
            }
            else
                p_ind[l * ind_pitch] = l + 1;
            max_dist = p_dist[curr_row];
        }

        // Part 2 : insert element in the k-th first lines
        max_row = (k - 1) * dist_pitch;
        for (l = k; l < height; l++)
        {
            curr_dist = p_dist[l * dist_pitch];
            if (curr_dist < max_dist)
            {
                i = k - 1;
                for (int a = 0; a < k - 1; a++)
                {
                    if (p_dist[a * dist_pitch] > curr_dist)
                    {
                        i = a;
                        break;
                    }
                }
                for (j = k - 1; j > i; j--)
                {
                    p_dist[j * dist_pitch] = p_dist[(j - 1) * dist_pitch];
                    p_ind[j * ind_pitch]   = p_ind[(j - 1) * ind_pitch];
                }
                p_dist[i * dist_pitch] = curr_dist;
                p_ind[i * ind_pitch]   = l + 1;
                max_dist             = p_dist[max_row];
            }
        }
    }
}


__global__ void cuAddQNormAndSqrt(float *dist, int width, int pitch, float *q, int k)
{
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;
    if (xIndex < width && yIndex < k)
        dist[yIndex * pitch + xIndex] = sqrt(dist[yIndex * pitch + xIndex] + q[xIndex]);
}



void printErrorMessage(hipError_t error, int memorySize)
{
    printf("==================================================\n");
    printf("MEMORY ALLOCATION ERROR  : %s\n", hipGetErrorString(error));
    printf("Whished allocated memory : %d\n", memorySize);
    printf("==================================================\n");
#if MATLAB_CODE == 1
    mexErrMsgTxt("CUDA ERROR DURING MEMORY ALLOCATION");
#endif
}

void knn(float *ref_host, int ref_width, float *query_host, int query_width, int height, int k, float *dist_host, int *ind_host)
{

    unsigned int size_of_float = sizeof(float);
    unsigned int size_of_int   = sizeof(int);

    // Variables
    float        *query_dev;
    float        *ref_dev;
    float        *dist_dev;
    float        *query_norm;
    float        *ref_norm;
    int          *ind_dev;
    hipError_t  result;
    size_t       query_pitch;
    size_t       query_pitch_in_bytes;
    size_t       ref_pitch;
    size_t       ref_pitch_in_bytes;
    size_t       ind_pitch;
    size_t       ind_pitch_in_bytes;
    size_t       max_nb_query_traited;
    size_t       actual_nb_query_width;
    size_t memory_total;
    size_t memory_free;

    // CUDA Initialisation
    hipInit(0);

    // Check free memory using driver API ; only (MAX_PART_OF_FREE_MEMORY_USED*100)% of memory will be used
    hipCtx_t cuContext;
    hipDevice_t  cuDevice = 0;
    hipCtxCreate(&cuContext, 0, cuDevice);
    hipMemGetInfo(&memory_free, &memory_total);
    cuCtxDetach (cuContext);

    // Determine maximum number of query that can be treated
    max_nb_query_traited = ( memory_free * MAX_PART_OF_FREE_MEMORY_USED - size_of_float * ref_width * (height + 1) ) / ( size_of_float * (height + ref_width + 1) + size_of_int * k);
    max_nb_query_traited = min( query_width, (max_nb_query_traited / 16) * 16 );

    // Allocation of global memory for query points and for distances
    result = hipMallocPitch( (void **) &query_dev, &query_pitch_in_bytes, max_nb_query_traited * size_of_float, height + ref_width + 1);
    if (result)
    {
        printErrorMessage(result, max_nb_query_traited * size_of_float * (height + ref_width));
        return;
    }
    query_pitch = query_pitch_in_bytes / size_of_float;
    query_norm  = query_dev  + height * query_pitch;
    dist_dev    = query_norm + query_pitch;

    // Allocation of global memory for reference points and ||query||
    result = hipMallocPitch((void **) &ref_dev, &ref_pitch_in_bytes, ref_width * size_of_float, height + 1);
    if (result)
    {
        printErrorMessage(result, ref_width * size_of_float * ( height + 1 ));
        hipFree(query_dev);
        return;
    }
    ref_pitch = ref_pitch_in_bytes / size_of_float;
    ref_norm  = ref_dev + height * ref_pitch;

    // Allocation of global memory for indexes
    result = hipMallocPitch( (void **) &ind_dev, &ind_pitch_in_bytes, max_nb_query_traited * size_of_int, k);
    if (result)
    {
        printErrorMessage(result, max_nb_query_traited * size_of_int * k);
        hipFree(ref_dev);
        hipFree(query_dev);
        return;
    }
    ind_pitch = ind_pitch_in_bytes / size_of_int;

    // Memory copy of ref_host in ref_dev
    result = hipMemcpy2D(ref_dev, ref_pitch_in_bytes, ref_host, ref_width * size_of_float, ref_width * size_of_float, height, hipMemcpyHostToDevice);

    // Computation of reference square norm
    dim3 ref_grid(ref_width / 256, 1, 1);
    dim3 ref_thread(256, 1, 1);
    if (ref_width % 256 != 0) ref_grid.x += 1;
    cuComputeNorm <<< ref_grid, ref_thread>>>(ref_dev, ref_width, ref_pitch, height, ref_norm);

    // Split queries to fit in GPU memory
    for (int i = 0; i < query_width; i += max_nb_query_traited)
    {

        // Number of query points considered
        actual_nb_query_width = min( max_nb_query_traited, query_width - i );

        // Copy of part of query actually being treated
        hipMemcpy2D(query_dev, query_pitch_in_bytes, &query_host[i], query_width * size_of_float, actual_nb_query_width * size_of_float, height, hipMemcpyHostToDevice);

        // Computation of Q square norm
        dim3 query_grid_1(actual_nb_query_width / 256, 1, 1);
        dim3 query_thread_1(256, 1, 1);
        if (actual_nb_query_width % 256 != 0) query_grid_1.x += 1;
        cuComputeNorm <<< query_grid_1, query_thread_1>>>(query_dev, actual_nb_query_width, query_pitch, height, query_norm);

        // Computation of Q*transpose(R)
        hipblasSgemm('n', 't', (int)query_pitch, (int)ref_pitch, height, (float) - 2.0, query_dev, query_pitch, ref_dev, ref_pitch, (float)0.0, dist_dev, query_pitch);

        // Add R norm to distances
        dim3 query_grid_2(actual_nb_query_width / 16, ref_width / 16, 1);
        dim3 query_thread_2(16, 16, 1);
        if (actual_nb_query_width % 16 != 0) query_grid_2.x += 1;
        if (ref_width % 16 != 0) query_grid_2.y += 1;
        cuAddRNorm <<< query_grid_2, query_thread_2>>>(dist_dev, actual_nb_query_width, query_pitch, ref_width, ref_norm);

        // Sort each column
        cuInsertionSort <<< query_grid_1, query_thread_1>>>(dist_dev, query_pitch, ind_dev, ind_pitch, actual_nb_query_width, ref_width, k);

        // Add Q norm and compute Sqrt ONLY ON ROW K-1
        cuAddQNormAndSqrt <<< query_grid_2, query_thread_2>>>( dist_dev, actual_nb_query_width, query_pitch, query_norm, k);

        // Memory copy of output from device to host
        hipMemcpy2D(&dist_host[i], query_width * size_of_float, dist_dev, query_pitch_in_bytes, actual_nb_query_width * size_of_float, k, hipMemcpyDeviceToHost);
        hipMemcpy2D(&ind_host[i],  query_width * size_of_int,   ind_dev,  ind_pitch_in_bytes,   actual_nb_query_width * size_of_int,   k, hipMemcpyDeviceToHost);
    }

    // Free memory
    hipFree(ind_dev);
    hipFree(ref_dev);
    hipFree(query_dev);
}




/**
  * Example of use of kNN search CUDA.
  */
int main(void)
{

    // Variables and parameters
    float *ref;                 // struct Pointer to reference point array
    float *query;               // struct Pointer to query point array
    float *dist;                // struct Pointer to distance array
    int   *ind;                 // struct Pointer to index array
    int    ref_nb     = 4096;   // Reference point number, max=65535
    int    query_nb   = 4096;   // Query point number,     max=65535
    int    dim        = 32;     // Dimension of points,    max=8192
    int    k          = 20;     // Nearest neighbors to consider
    int    iterations = 100;
    int    i;

    // Memory allocation
    ref    = (float *) malloc(ref_nb   * dim * sizeof(float));
    query  = (float *) malloc(query_nb * dim * sizeof(float));
    dist   = (float *) malloc(query_nb * k * sizeof(float));
    ind    = (int *)   malloc(query_nb * k * sizeof(float));

    // Init
    srand(time(NULL));
    for (i = 0 ; i < ref_nb   * dim ; i++) ref[i]    = (float)rand() / (float)RAND_MAX;
    for (i = 0 ; i < query_nb * dim ; i++) query[i]  = (float)rand() / (float)RAND_MAX;

    // Variables for duration evaluation
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed_time;

    // Display informations
    printf("Number of reference points      : %6d\n", ref_nb  );
    printf("Number of query points          : %6d\n", query_nb);
    printf("Dimension of points             : %4d\n", dim     );
    printf("Number of neighbors to consider : %4d\n", k       );
    printf("Processing kNN search           :"                );

    // Call kNN search CUDA
    hipEventRecord(start, 0);
    for (i = 0; i < iterations; i++)
        knn(ref, ref_nb, query, query_nb, dim, k, dist, ind);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf(" done in %f s for %d iterations (%f s by iteration)\n", elapsed_time / 1000, iterations, elapsed_time / (iterations * 1000));

    // Destroy cuda event object and free memory
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(ind);
    free(dist);
    free(query);
    free(ref);
}

