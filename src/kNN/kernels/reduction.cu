#include "hip/hip_runtime.h"
#include "reduction.cuh"
#include "hip/hip_runtime.h"
#include "stdio.h"



// #define SHARED_SIZE_LIMIT 8U
#define SHARED_SIZE_LIMIT 512U

// Can be optimized
__device__ int nearestPowerOf2 (int n)
{
  if (!n){
   return n;  //(0 == 2^0)
 }
 int x = 1;
 while(x <= n)
 {
  x <<= 1;
}
return x;
}

void compare(float &distA, int &indA, float &distB, int &indB, int dir)
{
  float f;
  int i;
  if ((distA  >= distB) == dir)
  {
    f = distA;
    distA  = distB;
    distB = f;
    i = indA;
    indA = indB;
    indB = i;
  }
}

__device__ void cuCompare_r(Distance &distA,  Distance &distB, int dir)
{
  Distance f;
  if ((distA.value  >= distB.value) == dir)
  {
    f = distA;
    distA  = distB;
    distB = f;
  }
}



__global__ void min_reduction(Distance *dist, int n, int threadOffset)
{

  int  thread1, halfPoint, index1,index2,offset;
  int threadOffset1 = max(1, threadOffset);
  int elements_in_block = nearestPowerOf2(n);
  offset = elements_in_block-n;
  dist += blockIdx.x*n;
  while(elements_in_block > 1)
  {
    thread1 = threadIdx.x;
    halfPoint = (elements_in_block / 2);
    while(thread1 < halfPoint)
    {
     if (thread1 + halfPoint   < elements_in_block-offset)
     {
      index1 =thread1 *threadOffset1;
      index2 = index1  + halfPoint * threadOffset1;
      cuCompare_r(dist[index1], dist[index2], 1);
    }
    thread1 +=blockDim.x;
  }
  __syncthreads();
  offset = 0;
  elements_in_block = halfPoint;
}
}

void knn_min_reduce(Distance* d_dist, int n){
  int blockCount, threadCount, elements_in_block, elements_out_of_block, offset;
  blockCount = ceil((float)n/SHARED_SIZE_LIMIT );
  elements_in_block = n/blockCount;
  if (blockCount == 0)
  {
    elements_in_block = n;
    blockCount = 1;
  }
  threadCount = elements_in_block /2;
  threadCount = min(SHARED_SIZE_LIMIT, threadCount);
  elements_out_of_block = n - blockCount * elements_in_block;
  if (elements_out_of_block > 0 )
  {
    offset = n - (elements_out_of_block *2);
    min_reduction<<<1,elements_out_of_block>>>(d_dist+offset+offset,elements_out_of_block*2,0);
  }
  min_reduction<<<blockCount,threadCount>>>(d_dist,elements_in_block,0);
  min_reduction<<<1,blockCount>>>(d_dist,blockCount,elements_in_block);
}



void min_reduce(float *h_dist, int *h_ind, int n){

  Distance *d_dist;
  int blockCount, threadCount, elements_in_block;



  blockCount = ceil((float)n/SHARED_SIZE_LIMIT );
  elements_in_block = n/blockCount;
  if (blockCount == 0)
  {
    elements_in_block = n;
    blockCount = 1;
  }
  threadCount = elements_in_block  /2;
  threadCount = min(SHARED_SIZE_LIMIT, threadCount);

  for (int i = n-1; i >= elements_in_block * blockCount; --i)
  {
    compare(h_dist[0], h_ind[0], h_dist[i], h_ind[i], 1);

  }
  hipMalloc( (void **) &d_dist, n* sizeof(Distance));

  hipMemcpy(d_dist,h_dist, n*sizeof(Distance), hipMemcpyHostToDevice);
  min_reduction<<<blockCount,threadCount>>>(d_dist,elements_in_block,0);
  min_reduction<<<1,blockCount>>>(d_dist,blockCount,elements_in_block);


  hipMemcpy(h_dist,d_dist, n*sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_dist);
}
