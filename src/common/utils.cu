#include "hip/hip_runtime_api.h"

#include "utils.cuh"
#include "knn_gpgpu.h"



void cuSetDevice(int device)
{
    checkCudaErrors(hipSetDevice(device));
}

int cuGetDevice()
{
    int device;
    checkCudaErrors(hipGetDevice(&device));
    return device;
}

int cuGetDeviceCount()
{
    int device_count;
    checkCudaErrors(hipGetDeviceCount(&device_count));
    return device_count;
}
