#include "hip/hip_runtime_api.h"

#include "utils.cuh"
#include "knn_gpgpu.h"


void cuSetDevice(int devive)
{
    checkCudaErrors(hipSetDevice(devive));
}
void cuGetDevice(int *devive)
{
    checkCudaErrors(hipGetDevice(devive));
}

void cuGetDeviceCount(int *device_count)
{
    checkCudaErrors(hipGetDeviceCount(device_count));
}

void hipStreamCreateWithFlags(hipStream_t *pStream, int device)
{
    int device_orig;
    cuGetDevice(&device_orig);
    cuSetDevice(device);
    checkCudaErrors(hipStreamCreate(pStream));
    cuSetDevice(device_orig);
}

void hipStreamSynchronize(hipStream_t stream)
{
    checkCudaErrors(hipStreamSynchronize(stream));
}

void hipStreamDestroy(hipStream_t stream)
{
    checkCudaErrors(hipStreamDestroy(stream));
}

void cuGetDeviceProperties(struct hipDeviceProp_t *prop, int device)
{
    hipGetDeviceProperties(prop, device);
}

