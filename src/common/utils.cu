#include "hip/hip_runtime_api.h"

#include "utils.cuh"
#include "knn_gpgpu.h"

void cuSetDevice(int device)
{
    checkCudaErrors(hipSetDevice(device));
}

int cuGetDevice()
{
    int device;
    checkCudaErrors(hipGetDevice(&device));
    return device;
}

int cuGetDeviceCount()
{
    int device_count;
    checkCudaErrors(hipGetDeviceCount(&device_count));
    return device_count;
}

size_t getFreeBytesOnGpu()
{
    size_t free_byte, total_byte ;
    hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
    return free_byte - 1024;
}
