#include "hip/hip_runtime_api.h"

#include "utils.cuh"
#include "knn_gpgpu.h"



void cuSetDevice(int devive)
{
    checkCudaErrors(hipSetDevice(devive));
}
